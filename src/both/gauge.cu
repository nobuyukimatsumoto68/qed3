#include "hip/hip_runtime.h"
#include <typeinfo>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cstdlib>
#include <cassert>

#include <algorithm>


#include <cstdint>
#include <complex>

using Double = double;
using Idx = std::int32_t;
using Complex = std::complex<double>;

// #define IS_DUAL
// #define IS_OVERLAP

// #define IsVerbose
// #define InfoForce
// #define InfoDelta

namespace Comp{
  constexpr bool is_compact=false;

#ifdef IS_OVERLAP
  constexpr int NPARALLEL=12; // 12
  constexpr int NPARALLEL2=1; // 12
  constexpr int NSTREAMS=4; // 4
#else
  constexpr int NPARALLEL=1; // 12
  constexpr int NPARALLEL2=12; // 12
  constexpr int NSTREAMS=12; // 4
#endif
  constexpr int NPARALLEL3=12; // 12

  constexpr int N_REFINE=4;
  constexpr int NS=2;

  constexpr Idx N_SITES=10*N_REFINE*N_REFINE+2;

  constexpr Idx N=NS*N_SITES; // matrix size of DW

  const double TOL_INNER=1.0e-9;
  const double TOL_OUTER=1.0e-8;
}

const std::string dir = "/mnt/hdd_barracuda/qed3/dats/";

#include "timer.h"

#include "s2n_simp.h"
#include "rng.h"
#include "gauge.h"
// #include "gauge_ext.h"
#include "action.h"
// #include "action_ext.h"

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hipsolver.h>
using CuC = hipDoubleComplex;
#include "gpu_header.h"

// ======================================

// #include "sparse_matrix.h"
// #include "dirac_simp.h"
// #include "sparse_dirac.h"
// #include "matpoly.h"
// #include "dirac_pf.h"
// #include "overlap.h"
// #include "pseudofermion.h"

# include "integrator.h"
#include "hmc.h"


// TODO: Cusparse for SparseMatrix::act_gpu, probably defining handle in matpoly.h
// make 2 streams in V Vdag in square in Overlap
// all the operation on GPU in Overlap::operator()
// pseudofermion
// 3d
// __m256 to vectorize with AVX2


int main(int argc, char* argv[]){
  std::cout << std::scientific << std::setprecision(15);
  std::clog << std::scientific << std::setprecision(15);

  int device;
  CUDA_CHECK(hipGetDeviceCount(&device));
  hipDeviceProp_t device_prop[device];
  hipGetDeviceProperties(&device_prop[0], 0);
  std::cout << "# dev = " << device_prop[0].name << std::endl;
  CUDA_CHECK(hipSetDevice(0));// "TITAN V"
  std::cout << "# (GPU device is set.)" << std::endl;

  // ---------------------------------------

  std::cout << "# N = " << Comp::N << std::endl;

  // --------------------
  using Link = std::array<Idx,2>; // <int,int>;
  constexpr Idx N = Comp::N;

  using Lattice=S2Simp;

  using Force=U1onS2<Lattice,false>;
  using Gauge=U1onS2<Lattice,false>;
  using Action=U1Wilson;

  using Rng=ParallelRng2<Lattice>;

  Lattice lattice(Comp::N_REFINE);
  std::cout << "# lattice set. " << std::endl;

  // ----------------------

  // const double gR = 10.0;
  double beta = 24.0; // 1.0/(gR*gR);
  if(argc==2) beta = atof( argv[1] );
  Action SW(beta);

  Gauge U(lattice);
  Rng rng(lattice);
  U.gaussian( rng, 0.2 );


  // //--------------------------------

  // int s = 0;
  // const Idx ix = 0;
  // const int yy=0;
  // Link link{ix, lattice.nns[ix][yy]};
  // std::cout << U.sp( s, link ) << std::endl
  //           << U.tp( s, link[1] ) << std::endl
  //           << U.sp( s+1, link[1] ) << std::endl
  //           << U.tp( s, link[0] ) << std::endl;
  // std::cout << "angle = " << U.plaquette_angle( s, link ) << std::endl;

  // std::cout << "debug. tp = " << std::endl;
  // for(const auto& v : U.temporal) {
  //   for(const double& elem : v) {
  //     std::cout << elem << " ";
  //   }
  //   std::cout << std::endl;
  // }

  // std::cout << "S(U) = " << SW(U) << std::endl;

  // Force grad(lattice);
  // SW.get_force( grad, U );


  // {  // sp
  //   for(int sl=0; sl<Comp::Nt; sl++){ // const int sl=0;
  //     for(Idx il=0; il<lattice.n_links; il++){ //Idx il=0;
  //       Link ell = lattice.links[il];
  //       // std::cout << "dS = " << grad.sp(sl, il) << std::endl;

  //       const double eps = 1.0e-5;

  //       Gauge UP(U);
  //       UP.sp(sl,il) += eps;

  //       Gauge UM(U);
  //       UM.sp(sl,il) -= eps;

  //       double sfp = SW(UP);
  //       double sfm = SW(UM);

  //       double chck = (sfp-sfm)/(2.0*eps);
  //       // std::cout << "check = " << chck << std::endl;
  //       std::cout << "diff = " << grad.sp(sl, il)-chck << std::endl;
  //       assert( std::abs(grad.sp(sl, il)-chck) < 1.0e-5 );
  //     }}
  // }


  // {  // tp
  //   // const int sl=0;
  //   // Idx ix=0;
  //   for(int sl=0; sl<Comp::Nt; sl++){ // const int sl=0;
  //     for(Idx ix=0; ix<lattice.n_sites; ix++){ //Idx ix=0;
  //       // std::cout << "dS = " << grad.tp(sl, ix) << std::endl;

  //       const double eps = 1.0e-5;

  //       Gauge UP(U);
  //       UP.tp(sl,ix) += eps;
  //       Gauge UM(U);
  //       UM.tp(sl,ix) -= eps;

  //       double sfp = SW(UP);
  //       double sfm = SW(UM);

  //       double chck = (sfp-sfm)/(2.0*eps);
  //       // std::cout << "check = " << chck << std::endl;
  //       std::cout << "diff = " << grad.tp(sl, ix)-chck << std::endl;
  //       assert( std::abs(grad.tp(sl, ix)-chck) < 1.0e-5 );
  //     }}
  // }


  // --------------------------------

  // Force pi(lattice);

  // HMCPureGauge hmc(rng, &SW, U, pi, 1.0, 10);

  // double rate, dH;
  // bool is_accept;
  // for(int k=0; k<10; k++){
  //   Timer timer;
  //   hmc.run( rate, dH, is_accept, true);
  //   std::cout << "# dH : " << dH
  //             << " is_accept : " << is_accept << std::endl;
  //   std::cout << "# HMC : " << timer.currentSeconds() << " sec" << std::endl;
  // }

  // pi.gaussian( rng );
  // Force pi0=pi;
  // Gauge U0=U;

  // const double tmax = 0.2; // 1.0; // 0.1
  // for(int nsteps=4; nsteps<=10; nsteps+=1){
  //   pi = pi0;
  //   U = U0;
  //   HMCPureGauge hmc(rng, &SW, U, pi, tmax, nsteps);
  //   const double h0 = hmc.H();
  //   hmc.integrate();
  //   const double h1 = hmc.H();
  //   double dH = h1-h0;
  //   std::cout << tmax/nsteps << " " << dH << std::endl;
  // }




  // --------------------------------


  //   using WilsonDirac=DiracS2Simp<Gauge>;

// #ifdef IS_OVERLAP
//   using Fermion=Overlap<Gauge,WilsonDirac,Lattice>;
// #else
//   using Fermion=DiracPf<Gauge,WilsonDirac,Lattice>;
// #endif

// #ifdef IS_OVERLAP
//   const double r = 1.0;
//   const double M5 = -1.6/2.0 * 0.5*(1.0 + std::sqrt( 5.0 + 2.0*std::sqrt(2.0) ));
// #else
//   const double r = 1.0;
//   const double M5 = 0.0;
// #endif
//   WilsonDirac DW(lattice, 0.0, r, M5);

//   std::cout << "# DW set" << std::endl;

//   Gauge U(lattice);
//   Rng rng(lattice);
//   // U.gaussian( rng, 0.2 );

//   // ---------------------

// #ifdef IS_OVERLAP
//   Fermion D(DW, 21);
//   std::cout << "# Dov set; M5 = " << M5 << std::endl;
//   D.update(U);
//   std::cout << "# min max ratio: "
//             << D.lambda_min << " "
//             << D.lambda_max << " "
//             << D.lambda_min/D.lambda_max << std::endl;
//   std::cout << "# delta = " << D.Delta() << std::endl;

//   auto f_DHD = std::bind(&Fermion::sq_deviceAsyncLaunch, &D,
//                          std::placeholders::_1, std::placeholders::_2);
//   auto f_DH = std::bind(&Fermion::adj_deviceAsyncLaunch, &D,
//                         std::placeholders::_1, std::placeholders::_2);
//   LinOpWrapper M_DHD( f_DHD );
//   // LinOpWrapper M_DH( f_DH );

//   // MatPoly DHD;
//   // DHD.push_back ( cplx(1.0), {&M_DHD} );
//   //
//   // MatPoly DH;
//   // DH.push_back ( cplx(1.0), {&M_DH} );
//   MatPoly Op_DHD; Op_DHD.push_back ( cplx(1.0), {&M_DHD} );
//   auto f_mgrad_DHD = std::bind(&Fermion::grad_deviceAsyncLaunch, &D,
//                                std::placeholders::_1, std::placeholders::_2, std::placeholders::_3);

// #else
//   Fermion D(DW);
//   // DWDevice<WilsonDirac,Lattice> d_DW(DW); // actual data used in M_DW, M_DWH
//   // CSR M_DW;
//   // CSR M_DWH;
//   // d_DW.associateCSR( M_DW, false );
//   // d_DW.associateCSR( M_DWH, true );
//   D.update( U );

//   auto f_DHD = std::bind(&Fermion::sq_deviceAsyncLaunch, &D,
//                          std::placeholders::_1, std::placeholders::_2);
//   auto f_DH = std::bind(&Fermion::adj_deviceAsyncLaunch, &D,
//                         std::placeholders::_1, std::placeholders::_2);

//   LinOpWrapper M_DHD( f_DHD );
//   MatPoly Op_DHD; Op_DHD.push_back ( cplx(1.0), {&M_DHD} );
//   // MatPoly Op_DHD;
//   // Op_DHD.push_back ( cplx(1.0), {&D.M_DW, &D.M_DWH} );
//   //
//   // MatPoly DH;
//   // DH.push_back ( cplx(1.0), {&D.M_DWH} );
//   // auto f_DHD = std::bind(&Fermion::sq_deviceAsyncLaunch, &Dov,
//   //                        std::placeholders::_1, std::placeholders::_2);
//   auto f_mgrad_DHD = std::bind(&Fermion::grad_deviceAsyncLaunch, &D,
//                                std::placeholders::_1, std::placeholders::_2, std::placeholders::_3);

// #endif


//   // -----------------------------------------------------------

//   const double gR = 0.4;
//   const double beta = 1.0/(gR*gR);
//   Action SW(beta);


//   PseudoFermion pf( Op_DHD, f_DH, f_mgrad_DHD, lattice );

//   // Timer timer;

//   // ------------------

//   // Idx il=1;
//   // Link ell = lattice.links[il];
//   // std::cout << "debug. ell = " << ell[0] << " " << ell[1] << std::endl;

//   // const double eps = 1.0e-5;
//   // Gauge UP(U);
//   // UP[il] += eps;
//   // Gauge UM(U);
//   // UM[il] -= eps;

//   // std::cout << " --- Dov.update : " << timer.currentSeconds() << std::endl;
//   // D.update(U);
//   // std::cout << " --- pf.gen : " << timer.currentSeconds() << std::endl;
//   // pf.gen( rng );

//   // std::cout << " --- grad constructor : " << timer.currentSeconds() << std::endl;
//   // Force grad(lattice);

//   // std::cout << " --- pre calc : " << timer.currentSeconds() << std::endl;
//   // D.precalc_grad_deviceAsyncLaunch( U, pf.d_eta );
//   // std::cout << " --- get force : " << timer.currentSeconds() << std::endl;
//   // pf.get_force( grad, U );

//   // std::cout << " --- fin : " << timer.currentSeconds() << std::endl;

//   // std::cout << "grad = " << grad[il] << std::endl;
//   // D.update(UP);
//   // pf.update_eta();
//   // double sfp = pf.S();

//   // D.update(UM);
//   // pf.update_eta();
//   // double sfm = pf.S();

//   // double chck = (sfp-sfm)/(2.0*eps);
//   // std::cout << "check = " << chck << std::endl;

//   // -----------------


//   // const double eps = 1.0e-5;

//   // std::cout << " --- Dov.update : " << timer.currentSeconds() << std::endl;
//   // Dov.update(U);
//   // std::cout << " --- pf.gen : " << timer.currentSeconds() << std::endl;
//   // pf.gen( rng );

//   // std::cout << " --- grad constructor : " << timer.currentSeconds() << std::endl;
//   // Force dSf(lattice);
//   // std::cout << " --- pre calc : " << timer.currentSeconds() << std::endl;
//   // Dov.precalc_grad_deviceAsyncLaunch( U, pf.d_eta );
//   // std::cout << " --- get force : " << timer.currentSeconds() << std::endl;
//   // pf.get_force( dSf, U );
//   // std::cout << " --- fin : " << timer.currentSeconds() << std::endl;


//   // for(Idx il=0; il<lattice.n_links; il++) std::cout << "grad = " << il << " " << dSf[il] << std::endl;

//   // const double tmax = 0.5; // 1.0; // 0.1
//   // const int nsteps=5;
//   // ExplicitLeapfrogML integrator( tmax, nsteps, 10 );


//   // Force pi( lattice );
//   // pi.gaussian( rng );
//   // // Force pi0=pi;

//   // for(Idx il=0; il<lattice.n_links; il++){
//   //   //   Idx il=3;
//   //   // Link ell = lattice.links[il];

//   //   Gauge UP(U);
//   //   UP[il] += eps;
//   //   Gauge UM(U);
//   //   UM[il] -= eps;


//   //   double Hp, Hm;
//   //   {
//   //     HMC hmc(rng, &SW, &Dov, UP, pi, &pf, &integrator);
//   //     Dov.update(UP);
//   //     pf.update_eta();
//   //     Hp = hmc.H();
//   //   }

//   //   {
//   //     HMC hmc(rng, &SW, &Dov, UM, pi, &pf, &integrator);
//   //     Dov.update(UM);
//   //     pf.update_eta();
//   //     Hm = hmc.H();
//   //   }

//   //   double chck = (Hp-Hm)/(2.0*eps);
//   //   std::cout << "check = " << il << " " << chck << std::endl;
//   // }
//   // // -----------------


//   // Force pi( lattice );
//   // pi.gaussian( rng );
//   // Force pi0=pi;

//   // Gauge U0=U;
//   // D.update(U);
//   // pf.gen( rng );
//   // D.precalc_grad_deviceAsyncLaunch( U, pf.d_eta );

//   // const double tmax = 0.2; // 1.0; // 0.1
//   // for(int nsteps=1; nsteps<=5; nsteps+=1){
//   //   // const int nsteps=5;
//   //   ExplicitLeapfrogML integrator( tmax, nsteps, 10 );
//   //   // ExplicitLeapfrogML integrator( tmax, nsteps, 100 );
//   //   pi = pi0;
//   //   U = U0;
//   //   HMC hmc(rng, &SW, &D, U, pi, &pf, &integrator);
//   //   D.update( U ); pf.update_eta();
//   //   D.precalc_grad_deviceAsyncLaunch( U, pf.d_eta );
//   //   const double h0 = hmc.H();
//   //   hmc.integrate();
//   //   const double h1 = hmc.H();
//   //   double dH = h1-h0;
//   //   std::cout << tmax/nsteps << " " << dH << std::endl;
//   // }



  Force pi(lattice);
  pi.gaussian( rng );
  Force pi0=pi;
  Gauge U0=U;

  const double tmax = 1.0; // 0.1
  const int nsteps=40;
  pi = pi0;
  U = U0;
  HMCPureGauge hmc(rng, &SW, U, pi, tmax, nsteps);

  double rate, dH;
  bool is_accept;
  for(int k=0; k<10; k++){
    Timer timer;
    hmc.run( rate, dH, is_accept, true );
    std::cout << "# dH : " << dH
              << " is_accept : " << is_accept << std::endl;
    // std::cout << "# HMC : " << timer.currentSeconds() << " sec" << std::endl;
  }
  for(int k=0; k<200; k++){
    Timer timer;
    hmc.run( rate, dH, is_accept );
    std::cout << "# dH : " << dH
              << " is_accept : " << is_accept << std::endl;
    // std::cout << "# HMC : " << timer.currentSeconds() << " sec" << std::endl;
  }



  std::vector<double> plaq_s0;
  std::vector<double> plaq_t0;
  const int il0 = 0;
  const int iface0 = 1;

  double r_mean;
  const int kmax=200;
  const int interval=10;

  for(int k=0; k<kmax; k++){
    Timer timer;
    hmc.run( rate, dH, is_accept);
    std::cout << "# dH : " << dH
              << " is_accept : " << is_accept << std::endl;
    r_mean += rate;

    if(k%interval==0){
      double tmp1 = 0.0;

      int counter1 = 0;
      for(int i_face=0; i_face<lattice.n_faces; i_face++){
        if( std::abs( lattice.vols[i_face]-lattice.vols[iface0] )>1.0e-10 ) continue;
        // if( std::abs( lattice.vols[i_face]-lattice.vols[iface0] )<1.0e-10 ) continue;
        counter1++;
        // tmp1[s] += std::cos( U.plaquette_angle(s, U.lattice.faces[i_face]) );
        // tmp1 += std::pow(lattice.mean_vol/lattice.vols[i_face], 1) * ( std::cos( U.plaquette_angle(U.lattice.faces[i_face]) ) - 1.0);
        tmp1 += std::pow(lattice.mean_vol/lattice.vols[i_face], 2) * std::pow( U.plaquette_angle(U.lattice.faces[i_face]), 2 );
      }
      tmp1 /= counter1;

      plaq_s0.push_back( tmp1 );
    }
    if(k%100==0){
      std::cout << "# k = " << k << std::endl;
    }
  }
  r_mean /= kmax;
  std::cout << "# r_mean = " << r_mean << std::endl;


  double mean_s0=0.0, var_s0=0.0;
  for(int k=0; k<plaq_s0.size(); k++){
    mean_s0 += plaq_s0[k];
  }
  mean_s0 /= plaq_s0.size();

  for(int k=0; k<plaq_s0.size(); k++){
    var_s0 += (plaq_s0[k]-mean_s0)*(plaq_s0[k]-mean_s0);
  }
  var_s0 /= plaq_s0.size()*plaq_s0.size();

  std::cout << "s0: " << iface0 << " "
            << beta << ", " << mean_s0 << ", " << std::sqrt( var_s0 ) << std::endl;
  // std::cout << "t0: " << beta << ", " << mean_t0 << ", " << std::sqrt( var_t0 ) << std::endl;
  // std::cout << "factor = " << U.lattice.mean_vol/U.lattice.vols[iface0] << std::endl;



  // CUDA_CHECK(hipDeviceReset());
  return 0;

}

