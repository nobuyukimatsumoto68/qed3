#include "hip/hip_runtime.h"
#include <typeinfo>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cstdlib>
#include <cassert>

#include <algorithm>


#include <cstdint>
#include <complex>

#include <array>
#include <vector>
#include <map>
#include <Eigen/Dense>

using Double = double;
using Idx = std::int32_t;
using Complex = std::complex<double>;

using MS=Eigen::Matrix2cd;
using VD=Eigen::Vector2d;
using VE=Eigen::Vector3d;
using VC=Eigen::VectorXcd;

static constexpr int NS = 2;
static constexpr int DIM = 2;
static constexpr Complex I = Complex(0.0, 1.0);


// #define IS_DUAL
// #define IS_OVERLAP

// #define IsVerbose
// #define InfoForce
// #define InfoDelta


namespace Comp{
  constexpr bool is_compact=false;

#ifdef IS_OVERLAP
  constexpr int NPARALLEL=12; // 12
  constexpr int NPARALLEL2=1; // 12
  constexpr int NSTREAMS=4; // 4
#else
  constexpr int NPARALLEL=1; // 12
  constexpr int NPARALLEL2=12; // 12
  constexpr int NSTREAMS=12; // 4
#endif
  constexpr int NPARALLEL3=10; // 12

  constexpr int N_REFINE=2;
  constexpr int NS=2;

  constexpr int Nt=4;

#ifdef IS_DUAL
  constexpr Idx N_SITES=20*N_REFINE*N_REFINE;
#else
  constexpr Idx N_SITES=10*N_REFINE*N_REFINE+2;
#endif

  constexpr Idx Nx=NS*N_SITES; // matrix size of DW
  constexpr Idx N=Nx*Nt; // matrix size of DW

  const double TOL_INNER=1.0e-9;
  const double TOL_OUTER=1.0e-8;
}

const std::string dir = "/mnt/hdd_barracuda/qed3/dats/";

#include "timer.h"

#include "s2n_simp.h"
#include "s2n_dual.h"
#include "rng.h"
#include "gauge_ext.h"
#include "action_ext.h"

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hipsolver.h>
using CuC = hipDoubleComplex;
#include "gpu_header.h"

// ======================================

#include "sparse_matrix.h"

#include "dirac_simp.h"
#include "dirac_dual.h"
#include "dirac_ext.h"

#include "sparse_dirac.h"
#include "matpoly.h"
#include "dirac_pf.h"
#include "overlap.h"
#include "pseudofermion.h"

# include "integrator.h"
#include "hmc.h"

// #include "obs.h" // to be developed



int main(int argc, char* argv[]){
  std::cout << std::scientific << std::setprecision(15);
  std::clog << std::scientific << std::setprecision(15);

  int device;
  CUDA_CHECK(hipGetDeviceCount(&device));
  hipDeviceProp_t device_prop[device];
  hipGetDeviceProperties(&device_prop[0], 0);
  std::cout << "# dev = " << device_prop[0].name << std::endl;
  CUDA_CHECK(hipSetDevice(0));// "TITAN V"
  std::cout << "# (GPU device is set.)" << std::endl;

  // ---------------------------------------
  using BaseLink = std::array<Idx,2>; // <int,int>;
  constexpr Idx N = Comp::N;
  constexpr int Nt = Comp::Nt;

#ifdef IS_DUAL
  using Base=S2Trivalent;
  using WilsonDirac=DiracExt<Base, DiracS2Dual>;
#else
  using Base=S2Simp;
  using WilsonDirac=DiracExt<Base, DiracS2Simp>;
#endif
  using Force=GaugeExt<Base,Nt,Comp::is_compact>;
  using Gauge=GaugeExt<Base,Nt,Comp::is_compact>;
  using Action=U1WilsonExt;
  using Rng=ParallelRngExt<Base,Nt>;


  Base base(Comp::N_REFINE);
  std::cout << "# lattice set. " << std::endl;

  // ----------------------

#ifdef IS_OVERLAP
  const double r = 1.0;
  const double M5 = -1.6/2.0 * 0.5*(1.0 + std::sqrt( 5.0 + 2.0*std::sqrt(2.0) ));
  using Fermion=Overlap<WilsonDirac>;
#else
  const double r = 1.0;
  const double M5 = 0.0;
  using Fermion=DiracPf<WilsonDirac>;
#endif
  const double c = 1.0;
  WilsonDirac DW(base, 0.0, 1.0, M5, c);


  std::cout << "# DW set" << std::endl;

  Gauge U(base);
  Rng rng(base);
  U.gaussian( rng, 0.2 );

  // ---------------------

#ifdef IS_OVERLAP
  Fermion D(DW, 31);
  std::cout << "# Dov set; M5 = " << M5 << std::endl;
  D.update(U);
  std::cout << "# min max ratio: "
            << D.lambda_min << " "
            << D.lambda_max << " "
            << D.lambda_min/D.lambda_max << std::endl;
  std::cout << "# delta = " << D.Delta() << std::endl;

  auto f_DHD = std::bind(&Fermion::sq_deviceAsyncLaunch, &D,
                         std::placeholders::_1, std::placeholders::_2);
  auto f_DH = std::bind(&Fermion::adj_deviceAsyncLaunch, &D,
                        std::placeholders::_1, std::placeholders::_2);

  LinOpWrapper M_DHD( f_DHD );
  MatPoly Op_DHD; Op_DHD.push_back ( cplx(1.0), {&M_DHD} );
#else
  Fermion D(DW);
  D.update( U );

  auto f_DHD = std::bind(&Fermion::sq_deviceAsyncLaunch, &D,
                         std::placeholders::_1, std::placeholders::_2);
  auto f_DH = std::bind(&Fermion::adj_deviceAsyncLaunch, &D,
                        std::placeholders::_1, std::placeholders::_2);

  LinOpWrapper M_DHD( f_DHD );
  MatPoly Op_DHD; Op_DHD.push_back ( cplx(1.0), {&M_DHD} );
#endif


  // -----------------------------------------------------------

  const double gR = 0.4;
  const double beta = 1.0/(gR*gR);
  Action SW(beta, beta);

  PseudoFermion pf( Op_DHD, f_DH, D, base );

  Timer timer;

  // ------------------

  // {
  //   int s=Nt-1;
  //   Idx il=4;
  //   BaseLink ell = base.links[il];
  //   std::cout << "debug. ell = " << ell[0] << " " << ell[1] << std::endl;

  //   const double eps = 1.0e-5;
  //   Gauge UP(U);
  //   UP.sp(s,il) += eps;
  //   Gauge UM(U);
  //   UM.sp(s,il) -= eps;

  //   std::cout << " --- Dov.update : " << timer.currentSeconds() << std::endl;
  //   D.update(U);
  //   std::cout << " --- pf.gen : " << timer.currentSeconds() << std::endl;
  //   pf.gen( rng );

  //   std::cout << " --- grad constructor : " << timer.currentSeconds() << std::endl;
  //   Force grad(base);

  //   std::cout << " --- pre calc : " << timer.currentSeconds() << std::endl;
  //   D.precalc_grad_deviceAsyncLaunch( U, pf.d_eta );
  //   std::cout << " --- get force : " << timer.currentSeconds() << std::endl;
  //   pf.get_force( grad, U );

  //   std::cout << " --- fin : " << timer.currentSeconds() << std::endl;

  //   std::cout << "grad = " << grad.sp(s,il) << std::endl;
  //   D.update(UP);
  //   pf.update_eta();
  //   double sfp = pf.S();

  //   D.update(UM);
  //   pf.update_eta();
  //   double sfm = pf.S();

  //   double chck = (sfp-sfm)/(2.0*eps);
  //   std::cout << "check = " << chck << std::endl;
  // }

  // // -----------------

  // {
  //   int s=Nt-1;
  //   Idx ix=4;

  //   const double eps = 1.0e-5;
  //   Gauge UP(U);
  //   UP.tp(s,ix) += eps;
  //   Gauge UM(U);
  //   UM.tp(s,ix) -= eps;

  //   std::cout << " --- Dov.update : " << timer.currentSeconds() << std::endl;
  //   D.update(U);
  //   std::cout << " --- pf.gen : " << timer.currentSeconds() << std::endl;
  //   pf.gen( rng );

  //   std::cout << " --- grad constructor : " << timer.currentSeconds() << std::endl;
  //   Force grad(base);

  //   std::cout << " --- pre calc : " << timer.currentSeconds() << std::endl;
  //   D.precalc_grad_deviceAsyncLaunch( U, pf.d_eta );
  //   std::cout << " --- get force : " << timer.currentSeconds() << std::endl;
  //   pf.get_force( grad, U );

  //   std::cout << " --- fin : " << timer.currentSeconds() << std::endl;

  //   std::cout << "grad = " << grad.tp(s, ix) << std::endl;
  //   D.update(UP);
  //   pf.update_eta();
  //   double sfp = pf.S();

  //   D.update(UM);
  //   pf.update_eta();
  //   double sfm = pf.S();

  //   double chck = (sfp-sfm)/(2.0*eps);
  //   std::cout << "check = " << chck << std::endl;
  // }

  // -----------------


  // const double eps = 1.0e-5;

  // std::cout << " --- D.update : " << timer.currentSeconds() << std::endl;
  // D.update(U);
  // std::cout << " --- pf.gen : " << timer.currentSeconds() << std::endl;
  // pf.gen( rng );

  // std::cout << " --- grad constructor : " << timer.currentSeconds() << std::endl;
  // Force dSf(base);
  // std::cout << " --- pre calc : " << timer.currentSeconds() << std::endl;
  // D.precalc_grad_deviceAsyncLaunch( U, pf.d_eta );
  // std::cout << " --- get force : " << timer.currentSeconds() << std::endl;
  // pf.get_force( dSf, U );
  // std::cout << " --- fin : " << timer.currentSeconds() << std::endl;


  // const double tmax = 0.5; // 1.0; // 0.1
  // const int nsteps=5;
  // ExplicitLeapfrogML integrator( tmax, nsteps, 10 );

  // // for(Idx il=0; il<base.n_links; il++) std::cout << "grad = " << il << " " << dSf[il] << std::endl;

  // Force pi( base );
  // pi.gaussian( rng );
  // // Force pi0=pi;

  // int s=2;
  // for(Idx il=0; il<base.n_links; il++){
  //   //   Idx il=3;
  //   // Link ell = base.links[il];

  //   Gauge UP(U);
  //   UP.sp(s,il) += eps;
  //   Gauge UM(U);
  //   UM.sp(s,il) -= eps;


  //   double Hp, Hm;
  //   {
  //     HMC hmc(rng, &SW, &D, UP, pi, &pf, &integrator);
  //     D.update(UP);
  //     pf.update_eta();
  //     Hp = hmc.H();
  //   }

  //   {
  //     HMC hmc(rng, &SW, &D, UM, pi, &pf, &integrator);
  //     D.update(UM);
  //     pf.update_eta();
  //     Hm = hmc.H();
  //   }

  //   double chck = (Hp-Hm)/(2.0*eps);
  //   std::cout << "check = " << il << " " << chck << " " << dSf.sp(s,il) << std::endl;
  // }

  // -----------------


  Force pi( base );
  pi.gaussian( rng );
  Force pi0=pi;

  Gauge U0=U;
  D.update(U);
  pf.gen( rng );
  D.precalc_grad_deviceAsyncLaunch( U, pf.d_eta );

  const double tmax = 0.2; // 1.0; // 0.1
  for(int nsteps=2; nsteps<=10; nsteps+=2){
    // const int nsteps=5;
    ExplicitLeapfrogML integrator( tmax, nsteps, 10 );
    // ExplicitLeapfrogML integrator( tmax, nsteps, 100 );
    pi = pi0;
    U = U0;
    HMC hmc(rng, &SW, &D, U, pi, &pf, &integrator);
    D.update( U ); pf.update_eta();
    D.precalc_grad_deviceAsyncLaunch( U, pf.d_eta );
    const double h0 = hmc.H();
    hmc.integrate();
    const double h1 = hmc.H();
    double dH = h1-h0;
    std::cout << tmax/nsteps << " " << dH << std::endl;
  }


  // -----------------


  // Force pi( base );
  // pi.gaussian( rng );
  // Force pi0=pi;

  // Gauge U0=U;
  // D.update(U);
  // pf.gen( rng );
  // D.precalc_grad_deviceAsyncLaunch( U, pf.d_eta );

  // const double tmax = 1.0; // 0.1
  // // for(int nsteps=1; nsteps<=5; nsteps+=1){
  // const int nsteps=8;
  // ExplicitLeapfrogML integrator( tmax, nsteps, 20 );
  // // ExplicitLeapfrogML integrator( tmax, nsteps, 100 );
  // pi = pi0;
  // U = U0;
  // HMC hmc(rng, &SW, &D, U, pi, &pf, &integrator);
  // D.update( U ); pf.update_eta();
  // D.precalc_grad_deviceAsyncLaunch( U, pf.d_eta );

  // double rate, dH;
  // bool is_accept;
  // for(int k=0; k<10; k++){
  //   Timer timer;
  //   hmc.run( rate, dH, is_accept, true);
  //   std::cout << "# dH : " << dH
  //             << " is_accept : " << is_accept << std::endl;
  //   std::cout << "# HMC : " << timer.currentSeconds() << " sec" << std::endl;
  // }

  // double r_mean;
  // const int kmax=50;
  // for(int k=0; k<kmax; k++){
  //   Timer timer;
  //   hmc.run( rate, dH, is_accept);
  //   std::cout << "# dH : " << dH
  //             << " is_accept : " << is_accept << std::endl;
  //   r_mean += rate;
  //   std::cout << "# HMC : " << timer.currentSeconds() << " sec" << std::endl;
  // }
  // r_mean /= kmax;
  // std::cout << "# r_mean = " << r_mean << std::endl;



  // CUDA_CHECK(hipDeviceReset());
  return 0;

}

