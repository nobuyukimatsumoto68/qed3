#include "hip/hip_runtime.h"
#include <typeinfo>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cstdlib>
#include <cassert>
#include <algorithm>
#include <cstdint>
#include <complex>
#include <array>
#include <vector>
#include <map>
#include <Eigen/Dense>

using Double = double;
using Idx = std::int32_t;
using Complex = std::complex<double>;

using Link = std::array<Idx,2>; // <int,int>;
using Face = std::vector<Idx>;

using MS=Eigen::Matrix2cd;
using VD=Eigen::Vector2d;
using VE=Eigen::Vector3d;
using VC=Eigen::VectorXcd;

static constexpr int NS = 2;
static constexpr int DIM = 2;
static constexpr Complex I = Complex(0.0, 1.0);


// #define IS_DUAL
// #define IS_OVERLAP
// #define IS_DAGGER
// #undef _OPENMP


namespace Comp{
  constexpr bool is_compact=false;

  // d_DW.update() is always done independently
#ifdef IS_OVERLAP
  constexpr int NPARALLEL_DUPDATE=1;
  constexpr int NPARALLEL=12; // 12
  constexpr int NSTREAMS=4; // 4
#else
  constexpr int NPARALLEL_DUPDATE=12;
  constexpr int NPARALLEL=1; // 12
  constexpr int NSTREAMS=12; // for grad loop
#endif
  constexpr int NPARALLEL_GAUGE=12; // 12
  constexpr int NPARALLEL_SORT=16; // 12

  constexpr int N_REFINE=4;
  constexpr int NS=2;

  constexpr int Nt=24;
  // constexpr int Nt=1;
  // constexpr int Nt=16;

#ifdef IS_DUAL
  constexpr Idx N_SITES=20*N_REFINE*N_REFINE;
#else
  constexpr Idx N_SITES=10*N_REFINE*N_REFINE+2;
#endif

  constexpr Idx Nx=NS*N_SITES; // matrix size of DW
  constexpr Idx N=Nx*Nt; // matrix size of DW

  const double TOL_INNER=1.0e-15;
  const double TOL_OUTER=1.0e-14;
}

const std::string dir = "/mnt/hdd_barracuda/qed3/dats/";

// // #define IsVerbose
// #define IsVerbose2
// // #define InfoForce
// #define InfoDelta

#include "timer.h"

#include "s2n_simp.h"
#include "s2n_dual.h"
#include "rng.h"
#include "gauge_ext.h"
#include "action_ext.h"

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hipsolver.h>
using CuC = hipDoubleComplex;
#include "gpu_header.h"

// ======================================

#include "sparse_matrix.h"
#include "dirac_base.h"
#include "dirac_simp.h"
#include "dirac_dual.h"
#include "dirac_ext.h"
// // #include "pseudofermion.h"
// #include "dirac.h"

#include "sparse_dirac.h"
#include "matpoly.h"

#include "dirac_pf.h"
#include "overlap.h"

#include "valence.h"

#include "../../integrator/geodesic.h"


// TODO: Cusparse for SparseMatrix::act_gpu, probably defining handle in matpoly.h
// make 2 streams in V Vdag in square in Overlap
// all the operation on GPU in Overlap::operator()
// pseudofermion
// 3d
// __m256 to vectorize with AVX2


int main(int argc, char* argv[]){
  std::cout << std::scientific << std::setprecision(15);
  std::clog << std::scientific << std::setprecision(15);

  int device;
  CUDA_CHECK(hipGetDeviceCount(&device));
  hipDeviceProp_t device_prop[device];
  hipGetDeviceProperties(&device_prop[0], 0);
  std::cout << "# dev = " << device_prop[0].name << std::endl;
  CUDA_CHECK(hipSetDevice(0));// "TITAN V"
  std::cout << "# (GPU device is set.)" << std::endl;

  // ---------------------------------------

  constexpr Idx N = Comp::N;
  constexpr int Nt = Comp::Nt;

#ifdef IS_DUAL
  using Base=S2Trivalent;
  using WilsonDirac=DiracExt<Base, DiracS2Dual>;
#else
  using Base=S2Simp;
  using WilsonDirac=DiracExt<Base, DiracS2Simp>;
#endif

  using Force=GaugeExt<Base,Nt,Comp::is_compact>;
  using Gauge=GaugeExt<Base,Nt,Comp::is_compact>;

  using Rng=ParallelRngExt<Base,Nt>;


  Base base(Comp::N_REFINE);
  std::cout << "# lattice set. " << std::endl;

  // ----------------------


  Gauge U(base);
  srand( time(NULL) );
  Rng rng(base, rand());

  // const double at = 0.5;
  const double T = 0.2;
  const double at = T/Comp::Nt;
  assert(std::sqrt(3.0)*base.mean_ell/at - 4.0/std::sqrt(3.0) > -1.0e-14);




  using Action=U1WilsonExt;
  const double gR = 10.0;
  double beta = 4.0; // 1.0/(gR*gR);
  Action SW(beta, at);

  U.gaussian( rng, 0.001 );

  std::cout << SW(U) << std::endl;

  U.random_gauge_trsf(rng, 2.0);
  U.random_gauge_trsf(rng, 2.0);
  U.random_gauge_trsf(rng, 2.0);

  std::cout << SW(U) << std::endl;







  return 1;





#ifdef IS_OVERLAP
  // Overlap Dov(DW, 31);
  // Dov.update(U);
  // std::cout << "# Dov set; M5 = " << M5 << std::endl;
  // std::cout << "# min max ratio: "
  //           << Dov.lambda_min << " "
  //           << Dov.lambda_max << " "
  //           << Dov.lambda_min/Dov.lambda_max << std::endl;
  // std::cout << "# delta = " << Dov.Delta() << std::endl;

  // auto f_Op = std::bind(&Overlap::mult_deviceAsyncLaunch, &Dov, std::placeholders::_1, std::placeholders::_2);
  // LinOpWrapper M_Op( f_Op );
  // Op.push_back ( cplx(1.0), {&M_Op} );

#ifdef IS_DUAL
  const double M5 = -1.5;
#else
  const double M5 = -1.0;
#endif

  WilsonDirac DW(base, 0.0, 1.0, M5, at);
  std::cout << "# DW set. " << std::endl;

  using Fermion=Overlap<WilsonDirac>;
  Fermion D(DW, 51);
  std::cout << "# D set. " << std::endl;
#else
  const double M5 = 0.0;
  WilsonDirac DW(base, 0.0, 1.0, M5, at);
  std::cout << "# DW set. " << std::endl;

  using Fermion=DiracPf<WilsonDirac>;
  Fermion D(DW);
  std::cout << "# D set. " << std::endl;
#endif

  D.update( U );
  std::cout << "# D updated. " << std::endl;

#ifdef IS_DAGGER
  auto f_pre = std::bind(&Fermion::mult_deviceAsyncLaunch, &D, std::placeholders::_1, std::placeholders::_2);
  auto f_sq = std::bind(&Fermion::DDH_deviceAsyncLaunch, &D, std::placeholders::_1, std::placeholders::_2);
#else
  auto f_pre = std::bind(&Fermion::adj_deviceAsyncLaunch, &D, std::placeholders::_1, std::placeholders::_2);
  auto f_sq = std::bind(&Fermion::DHD_deviceAsyncLaunch, &D, std::placeholders::_1, std::placeholders::_2);
#endif
  LinOpWrapper M_pre( f_pre );
  MatPoly pre; pre.push_back ( cplx(1.0), {&M_pre} );
  LinOpWrapper M_sq( f_sq );
  MatPoly sq; sq.push_back ( cplx(1.0), {&M_sq} );

  // ---------------------

  std::cout << "# calculating src " << std::endl;

  FermionVector src1; // (base, Nt, rng);
  FermionVector src; // (base, Nt, rng);
  src1.set_pt_source(0, 0, 0);
  // src1.set_pt_source(Comp::Nt/4, 0, 1);
  pre.from_cpu<N>( src.field, src1.field );

  FermionVector sink; // (base, Nt, rng);

  std::cout << "# calculating sink" << std::endl;

  sq.solve<N>( sink.field, src.field );

  std::cout << "# done" << std::endl;



  std::vector<double> thetas;
  std::vector<double> phis;
  std::vector<double> lengths;
#ifdef IS_DUAL
  {
    std::string dir = "/mnt/hdd_barracuda/qed3/dats/";
    std::vector<Geodesic::V3> sites;
    {
      std::ifstream file(dir+"pts_dual_n"+std::to_string(Comp::N_REFINE)+"_singlepatch.dat");

      std::string str;
      while (std::getline(file, str)){
        std::istringstream iss(str);
        double v1, v2, v3;
        iss >> v1;
        iss >> v2;
        iss >> v3;
        sites.push_back( Geodesic::V3(v1, v2, v3) );
      }
    }
    const auto x0 = sites[0];
    for(const auto& elem : sites){
      double len = Geodesic::geodesicLength(Geodesic::Pt(x0), Geodesic::Pt(elem));
      // std::cout << "len = " << len << std::endl;
      lengths.push_back(len);
      thetas.push_back( Geodesic::projectionS2(elem)[0] );
      phis.push_back( Geodesic::projectionS2(elem)[1] );
    }
  }
  // double alat;
  // {
  //   std::string dir = "/mnt/hdd_barracuda/qed3/dats/";
  //   std::ifstream file(dir+"alat_n"+std::to_string(Comp::N_REFINE)+"_singlepatch.dat");

  //   std::string str;
  //   std::getline(file, str);
  //   std::istringstream iss(str);
  //   iss >> alat;
  // }
#else
  {
    const auto x0 = base.sites[0];
    for(int ix=0; ix<base.n_sites; ix++){
      const auto x1 = base.sites[ix];
      double len = Geodesic::geodesicLength(Geodesic::Pt(x0), Geodesic::Pt(x1));
      // std::cout << "len = " << len << std::endl;
      lengths.push_back(len);
      thetas.push_back( Geodesic::projectionS2(x1)[0] );
      phis.push_back( Geodesic::projectionS2(x1)[1] );
    }
  }
#endif

  const double width = 0.05;

  double factor = at*base.mean_ell;
  if(Comp::Nt==1) factor = base.mean_ell;

  {
    std::string path = "prop_spacial_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+".dat1";
#ifdef IS_DUAL
    path = "dual_"+path;
#endif
#ifdef IS_OVERLAP
    path = "ov_"+path;
#endif
#ifdef IS_DAGGER
    path = "dagger_"+path;
#endif
    std::ofstream ofs(path);

    // Idx counter=0;
    for(Idx ix=0; ix<base.n_sites; ix++) {
      if( phis[ix]>width || phis[ix]<0. ) continue;
      {
        const auto elem = sink(0,ix,0);
        ofs << std::setw(25) << thetas[ix] << " "
          // ofs << std::setw(25) << lengths[ix] << " "
            // << std::setw(25) << 1.0/std::pow(base.mean_ell,2) * elem.real() << " "
            // << std::setw(25) << 1.0/std::pow(base.mean_ell,2) * elem.imag() << std::endl;
        << std::setw(25) << 1.0 * elem.real() / factor << " "
        << std::setw(25) << 1.0 * elem.imag() / factor << std::endl;
      }
      {
        const auto elem = sink(0,ix,1);
        ofs << std::setw(25) << thetas[ix] << " "
          // ofs << std::setw(25) << lengths[ix] << " "
            // << std::setw(25) << 1.0/std::pow(base.mean_ell,2) * elem.real() << " "
            // << std::setw(25) << 1.0/std::pow(base.mean_ell,2) * elem.imag() << std::endl;
            << std::setw(25) << 1.0 * elem.real() / factor << " "
            << std::setw(25) << 1.0 * elem.imag() / factor << std::endl;
      }
      // counter++;
    }
  }
  {
    std::string path = "prop_temporal_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+".dat1";
#ifdef IS_DUAL
    path = "dual_"+path;
#endif
#ifdef IS_OVERLAP
    path = "ov_"+path;
#endif
#ifdef IS_DAGGER
    path = "dagger_"+path;
#endif
    std::ofstream ofs(path);

    // Idx counter=0;
    for(Idx s=0; s<Comp::Nt; s++) {
      {
        const auto elem = sink(s,0,0);
        ofs << std::setw(25) << at*s << " "
          // ofs << std::setw(25) << s << " "
            // << std::setw(25) << 1.0/std::pow(base.mean_ell,2) * elem.real() << " "
            // << std::setw(25) << 1.0/std::pow(base.mean_ell,2) * elem.imag() << std::endl;
        << std::setw(25) << 1.0 * elem.real() / factor << " "
        << std::setw(25) << 1.0 * elem.imag() / factor << std::endl;
      }
      {
        const auto elem = sink(s,0,1);
        ofs << std::setw(25) << at*s << " "
          // ofs << std::setw(25) << s << " "
          // << std::setw(25) << 1.0/std::pow(base.mean_ell,2) * elem.real() << " "
            // << std::setw(25) << 1.0/std::pow(base.mean_ell,2) * elem.imag() << std::endl;
        << std::setw(25) << 1.0 * elem.real() / factor << " "
        << std::setw(25) << 1.0 * elem.imag() / factor << std::endl;
      }
      // counter++;
    }
  }





  // ------------------


  return 0;

}

