#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cstdlib>
#include <cassert>
#include <algorithm>
#include <cstdint>
#include <complex>
#include <array>
#include <vector>
#include <map>
#include <Eigen/Dense>

using Double = double;
using Idx = std::int32_t;
using Complex = std::complex<double>;

using MS=Eigen::Matrix2cd;
using VD=Eigen::Vector2d;
using VE=Eigen::Vector3d;
using VC=Eigen::VectorXcd;

static constexpr int NS = 2;
static constexpr int DIM = 2;
static constexpr Complex I = Complex(0.0, 1.0);


// #define IS_DUAL
// #define IS_OVERLAP


namespace Comp{
  constexpr bool is_compact=false;

#ifdef IS_OVERLAP
  constexpr int NPARALLEL=12; // 12
  constexpr int NPARALLEL2=1; // 12
  constexpr int NSTREAMS=4; // 4
#else
  constexpr int NPARALLEL=1; // 12
  constexpr int NPARALLEL2=12; // 12
  constexpr int NSTREAMS=12; // 4
#endif
  constexpr int NPARALLEL3=2; // 12

  constexpr int N_REFINE=2;
  constexpr int NS=2;

  constexpr int Nt=4;

#ifdef IS_DUAL
  constexpr Idx N_SITES=20*N_REFINE*N_REFINE;
#else
  constexpr Idx N_SITES=10*N_REFINE*N_REFINE+2;
#endif

  constexpr Idx Nx=NS*N_SITES; // matrix size of DW
  constexpr Idx N=Nx*Nt; // matrix size of DW

  const double TOL_INNER=1.0e-9;
  const double TOL_OUTER=1.0e-8;
}

const std::string dir = "/mnt/hdd_barracuda/qed3/dats/";

// // #define IsVerbose
// #define IsVerbose2
// // #define InfoForce
// #define InfoDelta

#include "timer.h"

#include "s2n_simp.h"
#include "s2n_dual.h"
#include "rng.h"
// #include "gauge.h"
#include "gauge_ext.h"
// #include "action.h"
// #include "action_ext.h"

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hipsolver.h>
using CuC = hipDoubleComplex;
#include "gpu_header.h"

// ======================================

#include "sparse_matrix.h"
#include "dirac_base.h"
#include "dirac_simp.h"
#include "dirac_dual.h"
#include "dirac_ext.h"
// // #include "pseudofermion.h"
// #include "dirac.h"

#include "sparse_dirac.h"
#include "matpoly.h"

#include "dirac_pf.h"

#include "overlap.h"

// #include "hmc.h"
// #include "dirac_s2_dual.h"
// #include "header_cusolver.hpp"


#include "../../integrator/geodesic.h"

// TODO: Cusparse for SparseMatrix::act_gpu, probably defining handle in matpoly.h
// all the operation on GPU in Overlap::operator()
// gradient of Dov (Overlap class, in parallel to Dirac)
// pseudofermion
// 3d
// __m256 to vectorize with AVX2




using BaseLink = std::array<Idx,2>; // <int,int>;
using BaseFace = std::vector<Idx>;



int main(int argc, char* argv[]){
  std::cout << std::scientific << std::setprecision(15);
  std::clog << std::scientific << std::setprecision(15);

  int device;
  CUDA_CHECK(hipGetDeviceCount(&device));
  hipDeviceProp_t device_prop[device];
  hipGetDeviceProperties(&device_prop[0], 0);
  std::cout << "# dev = " << device_prop[0].name << std::endl;
  CUDA_CHECK(hipSetDevice(0));// "TITAN V"
  std::cout << "# (GPU device is set.)" << std::endl;

  // ---------------------------------------
  // using Link = std::array<Idx,2>; // <int,int>;
  constexpr Idx N = Comp::N;
  constexpr int Nt = Comp::Nt;

#ifdef IS_DUAL
  using Base=S2Trivalent;
  using WilsonDirac=DiracExt<Base, DiracS2Dual>;
#else
  using Base=S2Simp;
  using WilsonDirac=DiracExt<Base, DiracS2Simp>;
#endif

  using Force=GaugeExt<Base,Nt,Comp::is_compact>;
  using Gauge=GaugeExt<Base,Nt,Comp::is_compact>;
  // using Action=U1WilsonExt;

  using Rng=ParallelRngExt<Base,Nt>;
  using Overlap=Overlap<WilsonDirac>;
  using Fermion=DiracPf<WilsonDirac>;

  Base base(Comp::N_REFINE);
  std::cout << "# lattice set. " << std::endl;

  // using Gauge=U1onS2<false>;
  // // using Force=U1onS2<false>;
  // // using Action=U1Wilson;
  // // using Fermion=Dirac1fonS2;
  // // using HMC=HMC<Force,Gauge,Action,Fermion>;
  // // using Rng=ParallelRng;
  // using Lattice=S2Trivalent;
  // using Rng=ParallelRng<Lattice>;

  // ----------------------

  // const double gR = 10.0;
  // double beta = 4.0; // 1.0/(gR*gR);
  // Action SW(beta, beta);

  Gauge U(base);
  srand( time(NULL) );
  Rng rng(base, rand());
  // U.gaussian( rng, 0.2 );


#ifdef IS_OVERLAP
  const double r = 1.0;
#ifdef IS_DUAL
  // const double M5 = -1.6/2.0 * 0.5*3.0/2.0;
  const double M5 = -1.2;
#else
  // const double M5 = -1.6/2.0 * 0.5*(1.0 + std::sqrt( 5.0 + 2.0*std::sqrt(2.0) ));
  const double M5 = -1.5;
#endif
#else // if not overlap
  const double r = 1.0;
  const double M5 = 0.0;
#endif
  const double c = 1.0;
  WilsonDirac DW(base, 0.0, 1.0, M5, c);

  Fermion D(DW);
  D.update( U );

  COO gmfourth;
  DW.volume_matrix( gmfourth.en, -0.5 );
  gmfourth.do_it();


  MatPoly Op;
#ifdef IS_OVERLAP
  Overlap Dov(DW, 31);
  Dov.update(U);
  std::cout << "# Dov set; M5 = " << M5 << std::endl;
  std::cout << "# min max ratio: "
            << Dov.lambda_min << " "
            << Dov.lambda_max << " "
            << Dov.lambda_min/Dov.lambda_max << std::endl;
  std::cout << "# delta = " << Dov.Delta() << std::endl;

  auto f_Op = std::bind(&Overlap::mult_deviceAsyncLaunch, &Dov, std::placeholders::_1, std::placeholders::_2);
  LinOpWrapper M_Op( f_Op );
  Op.push_back ( cplx(1.0), {&M_Op} );
#else
  // DWDevice<WilsonDirac,Lattice> d_DW(DW); // actual data used in M_DW, M_DWH
  // CSR M_DW;
  // CSR M_DWH;
  // d_DW.associateCSR( M_DW, false );
  // d_DW.associateCSR( M_DWH, true );
  // d_DW.update( U );
  // Op.push_back ( cplx(1.0), {&M_DW} );
  // Op.push_back ( cplx(1.0), {&M_Op} );

  auto f_Op = std::bind(&Fermion::mult_deviceAsyncLaunch, &D, std::placeholders::_1, std::placeholders::_2);
  LinOpWrapper M_Op( f_Op );
  Op.push_back ( cplx(1.0), {&gmfourth, &M_Op, &gmfourth} );
#endif


  Eigen::MatrixXcd mat(N, N);
  {
    for(Idx i=0; i<N; i++){
      Eigen::VectorXcd e = Eigen::VectorXcd::Zero(N);
      e(i) = 1.0;
      std::vector<Complex> xi(e.data(), e.data()+N);
      std::vector<Complex> Dxi(N);

      Op.from_cpu<N>( Dxi, xi );

      mat.block(0,i,N,1) = Eigen::Map<Eigen::MatrixXcd>(Dxi.data(), N, 1);
      std::cout << "# i = " << i << " finished." << std::endl;
    }
  }



  // =========================================
  // cusolver
  hipsolverHandle_t handle = NULL;
  hipStream_t stream = NULL;
  hipsolverDnParams_t params = NULL;

  const int n = mat.cols(); // Number of rows (or columns) of matrix A.
  const int lda = n;

  CuC *A, *W;
  A = (CuC*)malloc(n*n*CD);
  W = (CuC*)malloc(n*CD);
  for(int j=0; j<n; j++) for(int i=0; i<n; i++) A[n*j+i] = cplx(mat(i,j));
  // for(int j=0; j<n; j++) for(int i=0; i<n; i++) A[n*j+i] = reinterpret_cast<CuC*>(&mat(i,j));
  // for(int j=0; j<n; j++) for(int i=0; i<n; i++) A[n*j+i] = cplxmat(i,j));
  for(int i=0; i<n; i++) W[i] = cplx(0.);

  CuC *d_A, *d_W, *d_VL, *d_VR;

  hipsolverEigMode_t jobvl = HIPSOLVER_EIG_MODE_NOVECTOR;
  hipsolverEigMode_t jobvr = HIPSOLVER_EIG_MODE_NOVECTOR;
  int ldvl = n;
  int ldvr = n;
  //
  int info = 0;
  int *d_info = nullptr;

  size_t workspaceInBytesOnDevice = 0; /* size of workspace */
  void *d_work = nullptr;              /* device workspace */
  size_t workspaceInBytesOnHost = 0;   /* size of workspace */
  void *h_work = nullptr;              /* host workspace for */

  /* step 1: create cusolver handle, bind a stream */
  CUSOLVER_CHECK(hipsolverDnCreate(&handle));
  CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  CUSOLVER_CHECK(hipsolverSetStream(handle, stream));
  CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

  CUDA_CHECK(hipMalloc( &d_A, CD * n*n ));
  CUDA_CHECK(hipMalloc( &d_W, CD * n ));
  CUDA_CHECK(hipMalloc( &d_VL, CD * n*n ));
  CUDA_CHECK(hipMalloc( &d_VR, CD * n*n ));
  // CUDA_CHECK(hipMalloc( &d_VL, CD * 0 ));
  // CUDA_CHECK(hipMalloc( &d_VR, CD * 0 ));
  CUDA_CHECK(hipMalloc( &d_info, sizeof(int)));

  CUDA_CHECK( hipMemcpy(d_A, A, CD*n*n, H2D) );

  // step 3: query working space of syevd
  // hipsolverEigMode_t jobvl = HIPSOLVER_EIG_MODE_NOVECTOR;
  // hipsolverEigMode_t jobvr = HIPSOLVER_EIG_MODE_VECTOR;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

  CUSOLVER_CHECK( cusolverDnXgeev_bufferSize( handle,
        				      params,
        				      jobvl,
        				      jobvr,
        				      n,
        				      HIP_C_64F,
        				      d_A, // device
        				      lda,
        				      HIP_C_64F,
        				      d_W, // Array holding the computed eigenvalues of A
        				      HIP_C_64F,
        				      d_VL,
        				      ldvl,
        				      HIP_C_64F,
        				      d_VR,
        				      ldvr,
        				      HIP_C_64F,
        				      &workspaceInBytesOnDevice,
        				      &workspaceInBytesOnHost)
        	  );

  CUDA_CHECK(hipMalloc( &d_work, workspaceInBytesOnDevice ) );
  h_work = malloc(workspaceInBytesOnHost);

  // step 4: compute spectrum
  CUSOLVER_CHECK( cusolverDnXgeev( handle,
        			   params,
        			   jobvl,
        			   jobvr,
        			   n,
        			   HIP_C_64F,
        			   d_A,
        			   lda,
        			   HIP_C_64F,
        			   d_W,
        			   HIP_C_64F,
        			   d_VL,
        			   ldvl,
        			   HIP_C_64F,
        			   d_VR,
        			   ldvr,
        			   HIP_C_64F,
        			   d_work, // void *bufferOnDevice,
        			   workspaceInBytesOnDevice,
        			   h_work, // void *bufferOnHost,
        			   workspaceInBytesOnHost,
        			   d_info)
        	  );

  // ---------------------------------------------

  CUDA_CHECK(hipMemcpy( W, d_W, CD*n, D2H) );
  CUDA_CHECK(hipMemcpy( &info, d_info, sizeof(int), D2H ));

  std::vector<std::complex<double>> vr(n*n);
  for(Idx i=0; i<N; i++) gmfourth(d_VL+i*N, d_VR+i*N);
  CUDA_CHECK(hipMemcpy( reinterpret_cast<CuC*>(vr.data()), d_VL, CD * n*n, D2H ));

  std::cout << "# info (0=success) = " << info << std::endl;
  assert( info==0 );

  // std::vector<double> res(n);
  // for(int i=0; i<n; i++) res[i] = real(W[i]);
  // std::sort(res.begin(), res.end());
  // for(int i=0; i<n; i++) std::cout << i << " "
  // 				   << res[i] << " "
  // 				   << Dov.sgn(res[i]) << std::endl;

  std::vector<double> thetas;
  std::vector<double> phis;
#ifdef IS_DUAL
  std::vector<double> lengths;
  {
    std::string dir = "/mnt/hdd_barracuda/qed3/dats/";
    std::vector<Geodesic::V3> sites;
    {
      std::ifstream file(dir+"pts_dual_n"+std::to_string(Comp::N_REFINE)+"_singlepatch.dat");

      std::string str;
      while (std::getline(file, str)){
        std::istringstream iss(str);
        double v1, v2, v3;
        iss >> v1;
        iss >> v2;
        iss >> v3;
        sites.push_back( Geodesic::V3(v1, v2, v3) );
      }
    }
    const auto x0 = sites[0];
    for(const auto& elem : sites){
      double len = Geodesic::geodesicLength(Geodesic::Pt(x0), Geodesic::Pt(elem));
      // std::cout << "len = " << len << std::endl;
      lengths.push_back(len);
      thetas.push_back( Geodesic::projectionS2(elem)[0] );
      phis.push_back( Geodesic::projectionS2(elem)[1] );
    }
  }
  // double alat;
  // {
  //   std::string dir = "/mnt/hdd_barracuda/qed3/dats/";
  //   std::ifstream file(dir+"alat_n"+std::to_string(Comp::N_REFINE)+"_singlepatch.dat");

  //   std::string str;
  //   std::getline(file, str);
  //   std::istringstream iss(str);
  //   iss >> alat;
  // }
#else
  std::vector<double> lengths;
  {
    const auto x0 = base.sites[0];
    for(int ix=0; ix<base.n_sites; ix++){
      const auto x1 = base.sites[ix];
      double len = Geodesic::geodesicLength(Geodesic::Pt(x0), Geodesic::Pt(x1));
      // std::cout << "len = " << len << std::endl;
      lengths.push_back(len);
      thetas.push_back( Geodesic::projectionS2(x1)[0] );
      phis.push_back( Geodesic::projectionS2(x1)[1] );
    }
  }
#endif

  const double width = 0.05;

  if(Nt==1){
    {
      std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_0p.dat";
#ifdef IS_DUAL
      path = "dual_"+path;
#endif
      std::ofstream ofs(path);
      for(Idx ix=0; ix<base.n_sites; ix++) {
        // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
        if( phis[ix]>width || phis[ix]<0. ) continue;
        ofs << std::setw(25) << thetas[ix] << " "
            << std::setw(25) << real(vr[2*ix]) << " "
            << std::setw(25) << imag(vr[2*ix]) << std::endl;
            // << std::setw(25) << real(vr[Comp::Nx*ix]) << " "
            // << std::setw(25) << imag(vr[Comp::Nx*ix]) << std::endl;
      }
    }

    {
      std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_0m.dat";
#ifdef IS_DUAL
      path = "dual_"+path;
#endif
      std::ofstream ofs(path);
      for(Idx ix=0; ix<base.n_sites; ix++) {
        // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
        if( phis[ix]>width || phis[ix]<0. ) continue;
        ofs << std::setw(25) << thetas[ix] << " "
            << std::setw(25) << real(vr[2*ix+1]) << " "
            << std::setw(25) << imag(vr[2*ix+1]) << std::endl;
                  // << std::setw(25) << real(vr[Comp::Nx*ix+1]) << " "
                  // << std::setw(25) << imag(vr[Comp::Nx*ix+1]) << std::endl;
      }
    }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_many.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<8*base.n_sites; ix++) {
//         ofs << std::setw(25) << thetas[ix%base.n_sites] << " "
//             << std::setw(25) << real(vr[2*ix]) << " "
//             << std::setw(25) << imag(vr[2*ix]) << std::endl;
//         // << std::setw(25) << real(vr[Comp::Nx*ix]) << " "
//         // << std::setw(25) << imag(vr[Comp::Nx*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_0m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[2*ix+1]) << " "
//             << std::setw(25) << imag(vr[2*ix+1]) << std::endl;
//         // << std::setw(25) << real(vr[Comp::Nx*ix+1]) << " "
//         // << std::setw(25) << imag(vr[Comp::Nx*ix+1]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_1p.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[1*Comp::Nx+2*ix]) << " "
//             << std::setw(25) << imag(vr[1*Comp::Nx+2*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_1m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[1*Comp::Nx+2*ix+1]) << " "
//             << std::setw(25) << imag(vr[1*Comp::Nx+2*ix+1]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_2p.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[2*Comp::Nx+2*ix]) << " "
//             << std::setw(25) << imag(vr[2*Comp::Nx+2*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_2m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[2*Comp::Nx+2*ix+1]) << " "
//             << std::setw(25) << imag(vr[2*Comp::Nx+2*ix+1]) << std::endl;
//       }
//     }

//     // -----------------

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_4p.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//                   << std::setw(25) << real(vr[4*Comp::Nx+2*ix]) << " "
//                   << std::setw(25) << imag(vr[4*Comp::Nx+2*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_4m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//                   << std::setw(25) << real(vr[4*Comp::Nx+2*ix+1]) << " "
//                   << std::setw(25) << imag(vr[4*Comp::Nx+2*ix+1]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_5p.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[5*Comp::Nx+2*ix]) << " "
//             << std::setw(25) << imag(vr[5*Comp::Nx+2*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_5m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[5*Comp::Nx+2*ix+1]) << " "
//             << std::setw(25) << imag(vr[5*Comp::Nx+2*ix+1]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_6p.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//                   << std::setw(25) << real(vr[6*Comp::Nx+2*ix]) << " "
//                   << std::setw(25) << imag(vr[6*Comp::Nx+2*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_6m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//                   << std::setw(25) << real(vr[6*Comp::Nx+2*ix+1]) << " "
//                   << std::setw(25) << imag(vr[6*Comp::Nx+2*ix+1]) << std::endl;
//       }
//     }


//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_7p.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//                   << std::setw(25) << real(vr[7*Comp::Nx+2*ix]) << " "
//                   << std::setw(25) << imag(vr[7*Comp::Nx+2*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_7m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//                   << std::setw(25) << real(vr[7*Comp::Nx+2*ix+1]) << " "
//                   << std::setw(25) << imag(vr[7*Comp::Nx+2*ix+1]) << std::endl;
//       }
//     }


//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_8p.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//                   << std::setw(25) << real(vr[8*Comp::Nx+2*ix]) << " "
//                   << std::setw(25) << imag(vr[8*Comp::Nx+2*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_8m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//                   << std::setw(25) << real(vr[8*Comp::Nx+2*ix+1]) << " "
//                   << std::setw(25) << imag(vr[8*Comp::Nx+2*ix+1]) << std::endl;
//       }
//     }


//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_8p.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[8*Comp::Nx+2*ix]) << " "
//             << std::setw(25) << imag(vr[8*Comp::Nx+2*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_8m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[8*Comp::Nx+2*ix+1]) << " "
//             << std::setw(25) << imag(vr[8*Comp::Nx+2*ix+1]) << std::endl;
//       }
//     }

    // for(int i=0; i<n; i++) std::clog << real(vr[i]) << " " << imag(vr[i]) << std::endl;
  }

  for(int i=0; i<n; i++) std::clog << i << " " << real(W[i]) << " " << imag(W[i]) << " " << abs(W[i]) << std::endl;

  /* free resources */
  free(A);
  free(h_work);

  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_W));
  CUDA_CHECK(hipFree(d_VL));
  CUDA_CHECK(hipFree(d_VR));
  CUDA_CHECK(hipFree(d_info));
  CUDA_CHECK(hipFree(d_work));

  CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
  CUSOLVER_CHECK(hipsolverDnDestroy(handle));
  CUDA_CHECK(hipStreamDestroy(stream));


  return 0; // EXIT_SUCCESS;




  // auto f_DHD = std::bind(&Fermion::sq_deviceAsyncLaunch, &D,
  //                        std::placeholders::_1, std::placeholders::_2);
  // auto f_DH = std::bind(&Fermion::adj_deviceAsyncLaunch, &D,
  //                       std::placeholders::_1, std::placeholders::_2);

  // LinOpWrapper M_DHD( f_DHD );
  // MatPoly Op_DHD; Op_DHD.push_back ( cplx(1.0), {&M_DHD} );
  // auto f_mgrad_DHD = std::bind(&Fermion::grad_deviceAsyncLaunch, &D,
  //                              std::placeholders::_1, std::placeholders::_2, std::placeholders::_3);





  // Lattice lattice(Comp::N_REFINE);
  // // Dirac1fonS2 D(lattice, 0.0, 1.0);

  // using WilsonDirac=Dirac1fonS2;
  // // using Overlap=OverlapPseudoFermion;

  // Gauge U(lattice);
  // Rng rng(lattice);
  // U.gaussian( rng, 0.2 );

  // const double M5 = -1.8;
  // // const double M5 = 0.0;
  // // const double M5 = -2.5;
  // WilsonDirac DW(lattice, M5, 1.0/3.0);
  // // Overlap Dov(DW);
  // // Overlap Dov(DW, 1.0e-4, 21);
  // Overlap Dov(DW, 31);
  // // Dov.compute(U);
  // Dov.update(U);
  // std::cout << "# min max ratio: "
  //           << Dov.lambda_min << " "
  //           << Dov.lambda_max << " "
  //           << Dov.lambda_min/Dov.lambda_max << std::endl;
  // std::cout << "# delta = " << Dov.Delta() << std::endl;


  // MatPoly Op;
  // Op.push_back ( cplx(1.0), {&(Dov.M_DW), &(Dov.M_DWH)} );
  // auto f_Op = std::bind(&Overlap::sq_device, &Dov, std::placeholders::_1, std::placeholders::_2);
  // auto f_Op = std::bind(&Overlap::sq_device, &Dov, std::placeholders::_1, std::placeholders::_2);
  // auto f_Op = std::bind(&Overlap::mult_device, &Dov, std::placeholders::_1, std::placeholders::_2);
  // auto f_Op = std::bind(&Overlap::mult_deviceAsyncLaunch, &Dov, std::placeholders::_1, std::placeholders::_2);
  // LinOpWrapper M_Op( f_Op );

  // MatPoly Op;
  // Op.push_back ( cplx(1.0), {&M_Op} );
  // // Op.push_back ( cplx(1.0), {&Dov.M_DW} );

  // constexpr Idx N = Comp::N;
  // Eigen::MatrixXcd mat(N, N);
  // {
  //   for(Idx i=0; i<N; i++){
  //     Eigen::VectorXcd e = Eigen::VectorXcd::Zero(N);
  //     e(i) = 1.0;
  //     std::vector<Complex> xi(e.data(), e.data()+N);
  //     std::vector<Complex> Dxi(N);

  //     //   // Op.solve<N>( d_eta, d_xi );
  //     Op.from_cpu<N>( Dxi, xi );

  //     // for(Idx j=0; j<N; j++) Dxi[j] -= xi[j];
  //     // for(Idx j=0; j<N; j++) Dxi[j] -= M5*xi[j];
  //     // std::cout << "debug. i=" << i << std::endl;
  //     // Op.from_cpu<N>( Dxi, xi );
  //     mat.block(0,i,N,1) = Eigen::Map<Eigen::MatrixXcd>(Dxi.data(), N, 1);
  //     std::clog << "i = " << i << " finished." << std::endl;
  //   }
  // }

  // // =========================================
  // // cusolver
  // hipsolverHandle_t handle = NULL;
  // hipStream_t stream = NULL;
  // hipsolverDnParams_t params = NULL;

  // const int n = mat.cols(); // Number of rows (or columns) of matrix A.
  // const int lda = n;

  // CuC *A, *W;
  // A = (CuC*)malloc(n*n*CD);
  // W = (CuC*)malloc(n*CD);
  // for(int j=0; j<n; j++) for(int i=0; i<n; i++) A[n*j+i] = cplx(mat(i,j));
  // // for(int j=0; j<n; j++) for(int i=0; i<n; i++) A[n*j+i] = reinterpret_cast<CuC*>(&mat(i,j));
  // // for(int j=0; j<n; j++) for(int i=0; i<n; i++) A[n*j+i] = cplxmat(i,j));
  // for(int i=0; i<n; i++) W[i] = cplx(0.);

  // CuC *d_A, *d_W, *d_VL, *d_VR;
  // int ldvl = n;
  // int ldvr = n;
  // //
  // int info = 0;
  // int *d_info = nullptr;
  
  // size_t workspaceInBytesOnDevice = 0; /* size of workspace */
  // void *d_work = nullptr;              /* device workspace */
  // size_t workspaceInBytesOnHost = 0;   /* size of workspace */
  // void *h_work = nullptr;              /* host workspace for */

  // /* step 1: create cusolver handle, bind a stream */
  // CUSOLVER_CHECK(hipsolverDnCreate(&handle));
  // CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  // CUSOLVER_CHECK(hipsolverSetStream(handle, stream));
  // CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

  // CUDA_CHECK(hipMalloc( &d_A, CD * n*n ));
  // CUDA_CHECK(hipMalloc( &d_W, CD * n ));
  // CUDA_CHECK(hipMalloc( &d_VL, CD * n*n ));
  // CUDA_CHECK(hipMalloc( &d_VR, CD * n*n ));
  // CUDA_CHECK(hipMalloc( &d_info, sizeof(int)));

  // CUDA_CHECK( hipMemcpy(d_A, A, CD*n*n, H2D) );

  // // step 3: query working space of syevd
  // hipsolverEigMode_t jobvl = HIPSOLVER_EIG_MODE_NOVECTOR;
  // hipsolverEigMode_t jobvr = HIPSOLVER_EIG_MODE_NOVECTOR;
  // hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

  // CUSOLVER_CHECK( cusolverDnXgeev_bufferSize( handle,
  //       				      params,
  //       				      jobvl,
  //       				      jobvr,
  //       				      n,
  //       				      HIP_C_64F,
  //       				      d_A, // device
  //       				      lda,
  //       				      HIP_C_64F,
  //       				      d_W, // Array holding the computed eigenvalues of A
  //       				      HIP_C_64F,
  //       				      d_VL,
  //       				      ldvl,
  //       				      HIP_C_64F,
  //       				      d_VR,
  //       				      ldvr,
  //       				      HIP_C_64F,
  //       				      &workspaceInBytesOnDevice,
  //       				      &workspaceInBytesOnHost)
  //       	  );

  // CUDA_CHECK(hipMalloc( &d_work, workspaceInBytesOnDevice ) );
  // h_work = malloc(workspaceInBytesOnHost);

  // // step 4: compute spectrum
  // CUSOLVER_CHECK( cusolverDnXgeev( handle,
  //       			   params,
  //       			   jobvl,
  //       			   jobvr,
  //       			   n,
  //       			   HIP_C_64F,
  //       			   d_A,
  //       			   lda,
  //       			   HIP_C_64F,
  //       			   d_W,
  //       			   HIP_C_64F,
  //       			   d_VL,
  //       			   ldvl,
  //       			   HIP_C_64F,
  //       			   d_VR,
  //       			   ldvr,
  //       			   HIP_C_64F,
  //       			   d_work, // void *bufferOnDevice,
  //       			   workspaceInBytesOnDevice,
  //       			   h_work, // void *bufferOnHost,
  //       			   workspaceInBytesOnHost,
  //       			   d_info)
  //       	  );

  // // ---------------------------------------------

  // CUDA_CHECK(hipMemcpy( W, d_W, CD*n, D2H) );
  // CUDA_CHECK(hipMemcpy( &info, d_info, sizeof(int), D2H ));

  // std::cout << "# info (0=success) = " << info << std::endl;
  // assert( info==0 );

  // // std::vector<double> res(n);
  // // for(int i=0; i<n; i++) res[i] = real(W[i]);
  // // std::sort(res.begin(), res.end());
  // // for(int i=0; i<n; i++) std::cout << i << " "
  // // 				   << res[i] << " "
  // // 				   << Dov.sgn(res[i]) << std::endl;

  // for(int i=0; i<n; i++) std::cout << real(W[i]) << " " << imag(W[i]) << " " << abs(W[i]) << std::endl;

  // /* free resources */
  // free(A);
  // free(h_work);

  // CUDA_CHECK(hipFree(d_A));
  // CUDA_CHECK(hipFree(d_W));
  // CUDA_CHECK(hipFree(d_VL));
  // CUDA_CHECK(hipFree(d_VR));
  // CUDA_CHECK(hipFree(d_info));
  // CUDA_CHECK(hipFree(d_work));

  // CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
  // CUSOLVER_CHECK(hipsolverDnDestroy(handle));
  // CUDA_CHECK(hipStreamDestroy(stream));


  return 0; // EXIT_SUCCESS;

  // CUDA_CHECK(hipDeviceReset());



  // // 2.4.5.7. cusolverDnXgeev()
  // hipsolverStatus_t
  //   cusolverDnXgeev_bufferSize(
  // 			       hipsolverHandle_t handle,
  // 			       hipsolverDnParams_t params,
  // 			       hipsolverEigMode_t jobvl,
  // 			       hipsolverEigMode_t jobvr,
  // 			       int64_t n,
  // 			       hipDataType dataTypeA,
  // 			       const void *A,
  // 			       int64_t lda,
  // 			       hipDataType dataTypeW,
  // 			       const void *W,
  // 			       hipDataType dataTypeVL,
  // 			       const void *VL,
  // 			       int64_t ldvl,
  // 			       hipDataType dataTypeVR,
  // 			       const void *VR,
  // 			       int64_t ldvr,
  // 			       hipDataType computeType,
  // 			       size_t *workspaceInBytesOnDevice,
  // 			       size_t *workspaceInBytesOnHost);

  // // ss. 2.5.2.5. cusolverSp<t>csreigvsi()
  // cusolverSpZcsreigvsi(hipsolverSpHandle_t handle,
  // 		       int m,
  // 		       int nnz,
  // 		       const hipsparseMatDescr_t descrA,
  // 		       const cuDoubleCuC *csrValA,
  // 		       const int *csrRowPtrA,
  // 		       const int *csrColIndA,
  // 		       cuDoubleCuC mu0,
  // 		       const cuDoubleCuC *x0,
  // 		       int maxite,
  // 		       double tol,
  // 		       cuDoubleCuC *mu,
  // 		       cuDoubleCuC *x);

  // Eigen::CuCEigenSolver<Eigen::MatrixXcd> solver( mat );
  // const Eigen::MatrixXcd evec = solver.eigenvectors();
  // Eigen::VectorXcd ev = solver.eigenvalues();
  // for(int i=0; i<evec.rows(); i++){
  //   const Eigen::VectorXcd check1 = sq * evec.col(i);
  //   const Eigen::VectorXcd check2 = eval[i] * evec.col(i);
  //   assert( (check1-check2).norm() < 1.0e-8 );

  //   const Eigen::VectorXcd MV = mat * evec.col(i);
  //   std::cout << ( MV.array() / evec.col(i).array() - 1.0).abs().maxCoeff() << std::endl;
  // }

  // auto ev = mat.eigenvalues();
  // for(int i=0; i<ev.size(); i++){
  //   std::cout << ev[i].real() << " " << ev[i].imag() << std::endl;
  // }

  // ----------------------------------

    // return 0;
}



  // for(int ix=0; ix<lattice.n_sites; ix++){
  //   for(int jj=0; jj<lattice.sites[ix].nn; jj++){
  //     const int iy = lattice.sites[ix].neighbors[jj];
  //     auto mat1 = ( D.sigma[0] - D.gamma(ix, iy) ) * D.Omega(ix, iy);
  //     auto mat2 = D.Omega(ix, iy) * ( D.sigma[0] - D.gamma(iy, ix, M_PI) );
  //     std::cout << mat1-mat2 << std::endl;
  //   }}

  // ----------------------------------
