#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cstdlib>
#include <cassert>
#include <algorithm>
#include <cstdint>
#include <complex>
#include <array>
#include <vector>
#include <map>
#include <Eigen/Dense>

using Double = double;
using Idx = std::int32_t;
using Complex = std::complex<double>;

using MS=Eigen::Matrix2cd;
using VD=Eigen::Vector2d;
using VE=Eigen::Vector3d;
using VC=Eigen::VectorXcd;

static constexpr int NS = 2;
static constexpr int DIM = 2;
static constexpr Complex I = Complex(0.0, 1.0);


// #define IS_DUAL
// #define IS_OVERLAP

// // #define IsVerbose
// #define IsVerbose2
// // #define InfoForce
// #define InfoDelta



namespace Comp{
  constexpr bool is_compact=false;

  // d_DW.update() is always done independently
#ifdef IS_OVERLAP
  constexpr int NPARALLEL_DUPDATE=1;
  constexpr int NPARALLEL=12; // 12
  constexpr int NSTREAMS=4; // 4
#else
  constexpr int NPARALLEL_DUPDATE=12;
  constexpr int NPARALLEL=1; // 12
  constexpr int NSTREAMS=12; // for grad loop
#endif
  constexpr int NPARALLEL_GAUGE=12; // 12
  constexpr int NPARALLEL_SORT=12; // 12

  constexpr int N_REFINE=1;
  constexpr int NS=2;

  constexpr int Nt=3;

#ifdef IS_DUAL
  constexpr Idx N_SITES=20*N_REFINE*N_REFINE;
#else
  constexpr Idx N_SITES=10*N_REFINE*N_REFINE+2;
#endif

  constexpr Idx Nx=NS*N_SITES; // matrix size of DW
  constexpr Idx N=Nx*Nt; // matrix size of DW

  const double TOL_INNER=1.0e-9;
  const double TOL_OUTER=1.0e-8;
}

const std::string dir = "/mnt/hdd_barracuda/qed3/dats/";


#include "timer.h"

#include "s2n_simp.h"
#include "s2n_dual.h"
#include "rng.h"
#include "gauge_ext.h"

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hipsolver.h>
using CuC = hipDoubleComplex;
#include "gpu_header.h"

// ======================================

#include "sparse_matrix.h"
#include "dirac_base.h"
#include "dirac_simp.h"
#include "dirac_dual.h"
#include "dirac_ext.h"
// // #include "pseudofermion.h"
// #include "dirac.h"

#include "sparse_dirac.h"
#include "matpoly.h"

#include "dirac_pf.h"

#include "overlap.h"

// #include "hmc.h"
// #include "dirac_s2_dual.h"
// #include "header_cusolver.hpp"


#include "../../integrator/geodesic.h"

// TODO: Cusparse for SparseMatrix::act_gpu, probably defining handle in matpoly.h
// all the operation on GPU in Overlap::operator()
// gradient of Dov (Overlap class, in parallel to Dirac)
// pseudofermion
// 3d
// __m256 to vectorize with AVX2




using BaseLink = std::array<Idx,2>; // <int,int>;
using BaseFace = std::vector<Idx>;



int main(int argc, char* argv[]){
  std::cout << std::scientific << std::setprecision(15);
  std::clog << std::scientific << std::setprecision(15);

  int device;
  CUDA_CHECK(hipGetDeviceCount(&device));
  hipDeviceProp_t device_prop[device];
  hipGetDeviceProperties(&device_prop[0], 0);
  std::cout << "# dev = " << device_prop[0].name << std::endl;
  CUDA_CHECK(hipSetDevice(0));// "TITAN V"
  std::cout << "# (GPU device is set.)" << std::endl;

  // ---------------------------------------
  constexpr Idx N = Comp::N;
  constexpr int Nt = Comp::Nt;

#ifdef IS_DUAL
  using Base=S2Trivalent;
  using WilsonDirac=DiracExt<Base, DiracS2Dual>;
#else
  using Base=S2Simp;
  using WilsonDirac=DiracExt<Base, DiracS2Simp>;
#endif

  using Force=GaugeExt<Base,Nt,Comp::is_compact>;
  using Gauge=GaugeExt<Base,Nt,Comp::is_compact>;

  using Rng=ParallelRngExt<Base,Nt>;
  using Overlap=Overlap<WilsonDirac>;
  using Fermion=DiracPf<WilsonDirac>;

  Base base(Comp::N_REFINE);
  std::cout << "# lattice set. " << std::endl;

  // ----------------------

  // const double gR = 10.0;
  // double beta = 4.0; // 1.0/(gR*gR);
  // Action SW(beta, beta);

  Gauge U(base);
  srand( time(NULL) );
  Rng rng(base, rand());
  // U.gaussian( rng, 0.2 );


#ifdef IS_OVERLAP
  const double r = 1.0;
#ifdef IS_DUAL
  // const double M5 = -1.6/2.0 * 0.5*3.0/2.0;
  const double M5 = -1.2;
#else
  // const double M5 = -1.6/2.0 * 0.5*(1.0 + std::sqrt( 5.0 + 2.0*std::sqrt(2.0) ));
  // const double M5 = -1.5;
  const double M5 = -1.0;
#endif
#else // if not overlap
  const double r = 1.0;
  // const double r = 0.0;
  const double M5 = 0.0;
#endif
  // const double at = base.mean_ell * 1.0;
  const double at = 0.01;
  // const double at = 2.0/Comp::Nt;
  WilsonDirac DW(base, 0.0, r, M5, at);

  Fermion D(DW);
  D.update( U );

  COO gmfourth;
  DW.volume_matrix( gmfourth.en, -0.5 );
  gmfourth.do_it();


  MatPoly Op;
#ifdef IS_OVERLAP
  Overlap Dov(DW, 31);
  Dov.update(U);
  std::cout << "# Dov set; M5 = " << M5 << std::endl;
  std::cout << "# min max ratio: "
            << Dov.lambda_min << " "
            << Dov.lambda_max << " "
            << Dov.lambda_min/Dov.lambda_max << std::endl;
  std::cout << "# delta = " << Dov.Delta() << std::endl;

  auto f_Op = std::bind(&Overlap::mult_deviceAsyncLaunch, &Dov, std::placeholders::_1, std::placeholders::_2);
  LinOpWrapper M_Op( f_Op );
  Op.push_back ( cplx(1.0), {&M_Op} );
#else
  auto f_Op = std::bind(&Fermion::mult_deviceAsyncLaunch, &D, std::placeholders::_1, std::placeholders::_2);
  LinOpWrapper M_Op( f_Op );
  // Op.push_back ( cplx(1.0), {&gmfourth, &M_Op, &gmfourth} );
  Op.push_back ( cplx(1.0), {&M_Op} );
#endif


  Eigen::MatrixXcd mat(N, N);
  {
    for(Idx i=0; i<N; i++){
      Eigen::VectorXcd e = Eigen::VectorXcd::Zero(N);
      e(i) = 1.0;
      std::vector<Complex> xi(e.data(), e.data()+N);
      std::vector<Complex> Dxi(N);

      Op.from_cpu<N>( Dxi, xi );

      mat.block(0,i,N,1) = Eigen::Map<Eigen::MatrixXcd>(Dxi.data(), N, 1);
      std::cout << "# i = " << i << " finished." << std::endl;
    }
  }


  {
    // Eigen::IOFormat fmt(Eigen::FullPrecision, 0, ", ", ",\n", "{", "}", "{", "}");
    // Eigen::IOFormat CommaInitFmt(StreamPrecision, DontAlignCols, ", ", ", ", "", "", " << ", ";");
    std::clog << mat.real() << std::endl;
    std::clog << mat.imag() << std::endl;
    // std::clog << mat.real().format(fmt) << std::endl;
    // std::clog << mat.imag().format(fmt) << std::endl;
    return 0;
  }



  // =========================================
  // cusolver
  hipsolverHandle_t handle = NULL;
  hipStream_t stream = NULL;
  hipsolverDnParams_t params = NULL;

  const int n = mat.cols(); // Number of rows (or columns) of matrix A.
  const int lda = n;

  CuC *A, *W;
  A = (CuC*)malloc(n*n*CD);
  W = (CuC*)malloc(n*CD);
  for(int j=0; j<n; j++) for(int i=0; i<n; i++) A[n*j+i] = cplx(mat(i,j));
  for(int i=0; i<n; i++) W[i] = cplx(0.);

  CuC *d_A, *d_W, *d_VL, *d_VR;

  hipsolverEigMode_t jobvl = HIPSOLVER_EIG_MODE_NOVECTOR;
  hipsolverEigMode_t jobvr = HIPSOLVER_EIG_MODE_NOVECTOR;
  int ldvl = n;
  int ldvr = n;
  //
  int info = 0;
  int *d_info = nullptr;

  size_t workspaceInBytesOnDevice = 0; /* size of workspace */
  void *d_work = nullptr;              /* device workspace */
  size_t workspaceInBytesOnHost = 0;   /* size of workspace */
  void *h_work = nullptr;              /* host workspace for */

  /* step 1: create cusolver handle, bind a stream */
  CUSOLVER_CHECK(hipsolverDnCreate(&handle));
  CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  CUSOLVER_CHECK(hipsolverSetStream(handle, stream));
  CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

  CUDA_CHECK(hipMalloc( &d_A, CD * n*n ));
  CUDA_CHECK(hipMalloc( &d_W, CD * n ));
  CUDA_CHECK(hipMalloc( &d_VL, CD * n*n ));
  CUDA_CHECK(hipMalloc( &d_VR, CD * n*n ));
  // CUDA_CHECK(hipMalloc( &d_VL, CD * 0 ));
  // CUDA_CHECK(hipMalloc( &d_VR, CD * 0 ));
  CUDA_CHECK(hipMalloc( &d_info, sizeof(int)));

  CUDA_CHECK( hipMemcpy(d_A, A, CD*n*n, H2D) );

  // step 3: query working space of syevd
  // hipsolverEigMode_t jobvl = HIPSOLVER_EIG_MODE_NOVECTOR;
  // hipsolverEigMode_t jobvr = HIPSOLVER_EIG_MODE_VECTOR;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

  CUSOLVER_CHECK( cusolverDnXgeev_bufferSize( handle,
        				      params,
        				      jobvl,
        				      jobvr,
        				      n,
        				      HIP_C_64F,
        				      d_A, // device
        				      lda,
        				      HIP_C_64F,
        				      d_W, // Array holding the computed eigenvalues of A
        				      HIP_C_64F,
        				      d_VL,
        				      ldvl,
        				      HIP_C_64F,
        				      d_VR,
        				      ldvr,
        				      HIP_C_64F,
        				      &workspaceInBytesOnDevice,
        				      &workspaceInBytesOnHost)
        	  );

  CUDA_CHECK(hipMalloc( &d_work, workspaceInBytesOnDevice ) );
  h_work = malloc(workspaceInBytesOnHost);

  // step 4: compute spectrum
  CUSOLVER_CHECK( cusolverDnXgeev( handle,
        			   params,
        			   jobvl,
        			   jobvr,
        			   n,
        			   HIP_C_64F,
        			   d_A,
        			   lda,
        			   HIP_C_64F,
        			   d_W,
        			   HIP_C_64F,
        			   d_VL,
        			   ldvl,
        			   HIP_C_64F,
        			   d_VR,
        			   ldvr,
        			   HIP_C_64F,
        			   d_work, // void *bufferOnDevice,
        			   workspaceInBytesOnDevice,
        			   h_work, // void *bufferOnHost,
        			   workspaceInBytesOnHost,
        			   d_info)
        	  );

  // ---------------------------------------------

  CUDA_CHECK(hipMemcpy( W, d_W, CD*n, D2H) );
  CUDA_CHECK(hipMemcpy( &info, d_info, sizeof(int), D2H ));

  std::vector<std::complex<double>> vr(n*n);
  for(Idx i=0; i<N; i++) gmfourth(d_VL+i*N, d_VR+i*N);
  CUDA_CHECK(hipMemcpy( reinterpret_cast<CuC*>(vr.data()), d_VL, CD * n*n, D2H ));

  std::cout << "# info (0=success) = " << info << std::endl;
  assert( info==0 );

  // std::vector<double> res(n);
  // for(int i=0; i<n; i++) res[i] = real(W[i]);
  // std::sort(res.begin(), res.end());
  // for(int i=0; i<n; i++) std::cout << i << " "
  // 				   << res[i] << " "
  // 				   << Dov.sgn(res[i]) << std::endl;

  std::vector<double> thetas;
  std::vector<double> phis;
#ifdef IS_DUAL
  std::vector<double> lengths;
  {
    std::string dir = "/mnt/hdd_barracuda/qed3/dats/";
    std::vector<Geodesic::V3> sites;
    {
      std::ifstream file(dir+"pts_dual_n"+std::to_string(Comp::N_REFINE)+"_singlepatch.dat");

      std::string str;
      while (std::getline(file, str)){
        std::istringstream iss(str);
        double v1, v2, v3;
        iss >> v1;
        iss >> v2;
        iss >> v3;
        sites.push_back( Geodesic::V3(v1, v2, v3) );
      }
    }
    const auto x0 = sites[0];
    for(const auto& elem : sites){
      double len = Geodesic::geodesicLength(Geodesic::Pt(x0), Geodesic::Pt(elem));
      // std::cout << "len = " << len << std::endl;
      lengths.push_back(len);
      thetas.push_back( Geodesic::projectionS2(elem)[0] );
      phis.push_back( Geodesic::projectionS2(elem)[1] );
    }
  }
  // double alat;
  // {
  //   std::string dir = "/mnt/hdd_barracuda/qed3/dats/";
  //   std::ifstream file(dir+"alat_n"+std::to_string(Comp::N_REFINE)+"_singlepatch.dat");

  //   std::string str;
  //   std::getline(file, str);
  //   std::istringstream iss(str);
  //   iss >> alat;
  // }
#else
  std::vector<double> lengths;
  {
    const auto x0 = base.sites[0];
    for(int ix=0; ix<base.n_sites; ix++){
      const auto x1 = base.sites[ix];
      double len = Geodesic::geodesicLength(Geodesic::Pt(x0), Geodesic::Pt(x1));
      // std::cout << "len = " << len << std::endl;
      lengths.push_back(len);
      thetas.push_back( Geodesic::projectionS2(x1)[0] );
      phis.push_back( Geodesic::projectionS2(x1)[1] );
    }
  }
#endif

  const double width = 0.05;

  if(Nt==1){
    {
      std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_0p.dat";
#ifdef IS_DUAL
      path = "dual_"+path;
#endif
      std::ofstream ofs(path);
      for(Idx ix=0; ix<base.n_sites; ix++) {
        // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
        if( phis[ix]>width || phis[ix]<0. ) continue;
        ofs << std::setw(25) << thetas[ix] << " "
            << std::setw(25) << real(vr[2*ix]) << " "
            << std::setw(25) << imag(vr[2*ix]) << std::endl;
            // << std::setw(25) << real(vr[Comp::Nx*ix]) << " "
            // << std::setw(25) << imag(vr[Comp::Nx*ix]) << std::endl;
      }
    }

    {
      std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_0m.dat";
#ifdef IS_DUAL
      path = "dual_"+path;
#endif
      std::ofstream ofs(path);
      for(Idx ix=0; ix<base.n_sites; ix++) {
        // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
        if( phis[ix]>width || phis[ix]<0. ) continue;
        ofs << std::setw(25) << thetas[ix] << " "
            << std::setw(25) << real(vr[2*ix+1]) << " "
            << std::setw(25) << imag(vr[2*ix+1]) << std::endl;
                  // << std::setw(25) << real(vr[Comp::Nx*ix+1]) << " "
                  // << std::setw(25) << imag(vr[Comp::Nx*ix+1]) << std::endl;
      }
    }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_many.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<8*base.n_sites; ix++) {
//         ofs << std::setw(25) << thetas[ix%base.n_sites] << " "
//             << std::setw(25) << real(vr[2*ix]) << " "
//             << std::setw(25) << imag(vr[2*ix]) << std::endl;
//         // << std::setw(25) << real(vr[Comp::Nx*ix]) << " "
//         // << std::setw(25) << imag(vr[Comp::Nx*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_0m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[2*ix+1]) << " "
//             << std::setw(25) << imag(vr[2*ix+1]) << std::endl;
//         // << std::setw(25) << real(vr[Comp::Nx*ix+1]) << " "
//         // << std::setw(25) << imag(vr[Comp::Nx*ix+1]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_1p.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[1*Comp::Nx+2*ix]) << " "
//             << std::setw(25) << imag(vr[1*Comp::Nx+2*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_1m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[1*Comp::Nx+2*ix+1]) << " "
//             << std::setw(25) << imag(vr[1*Comp::Nx+2*ix+1]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_2p.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[2*Comp::Nx+2*ix]) << " "
//             << std::setw(25) << imag(vr[2*Comp::Nx+2*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_2m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[2*Comp::Nx+2*ix+1]) << " "
//             << std::setw(25) << imag(vr[2*Comp::Nx+2*ix+1]) << std::endl;
//       }
//     }

//     // -----------------

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_4p.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//                   << std::setw(25) << real(vr[4*Comp::Nx+2*ix]) << " "
//                   << std::setw(25) << imag(vr[4*Comp::Nx+2*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_4m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//                   << std::setw(25) << real(vr[4*Comp::Nx+2*ix+1]) << " "
//                   << std::setw(25) << imag(vr[4*Comp::Nx+2*ix+1]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_5p.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[5*Comp::Nx+2*ix]) << " "
//             << std::setw(25) << imag(vr[5*Comp::Nx+2*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_5m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[5*Comp::Nx+2*ix+1]) << " "
//             << std::setw(25) << imag(vr[5*Comp::Nx+2*ix+1]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_6p.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//                   << std::setw(25) << real(vr[6*Comp::Nx+2*ix]) << " "
//                   << std::setw(25) << imag(vr[6*Comp::Nx+2*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_6m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//                   << std::setw(25) << real(vr[6*Comp::Nx+2*ix+1]) << " "
//                   << std::setw(25) << imag(vr[6*Comp::Nx+2*ix+1]) << std::endl;
//       }
//     }


//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_7p.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//                   << std::setw(25) << real(vr[7*Comp::Nx+2*ix]) << " "
//                   << std::setw(25) << imag(vr[7*Comp::Nx+2*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_7m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//                   << std::setw(25) << real(vr[7*Comp::Nx+2*ix+1]) << " "
//                   << std::setw(25) << imag(vr[7*Comp::Nx+2*ix+1]) << std::endl;
//       }
//     }


//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_8p.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//                   << std::setw(25) << real(vr[8*Comp::Nx+2*ix]) << " "
//                   << std::setw(25) << imag(vr[8*Comp::Nx+2*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_8m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         // if( !Geodesic::isModdable(phis[ix], 2.0*M_PI, 0.1) ) continue;
//         if( phis[ix]>width || phis[ix]<0. ) continue;
//         ofs << std::setw(25) << thetas[ix] << " "
//                   << std::setw(25) << real(vr[8*Comp::Nx+2*ix+1]) << " "
//                   << std::setw(25) << imag(vr[8*Comp::Nx+2*ix+1]) << std::endl;
//       }
//     }


//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_8p.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[8*Comp::Nx+2*ix]) << " "
//             << std::setw(25) << imag(vr[8*Comp::Nx+2*ix]) << std::endl;
//       }
//     }

//     {
//       std::string path = "wf_L"+std::to_string(Comp::N_REFINE)+"_Nt"+std::to_string(Nt)+"_8m.dat";
// #ifdef IS_DUAL
//       path = "dual_"+path;
// #endif
//       std::ofstream ofs(path);
//       for(Idx ix=0; ix<base.n_sites; ix++) {
//         ofs << std::setw(25) << thetas[ix] << " "
//             << std::setw(25) << real(vr[8*Comp::Nx+2*ix+1]) << " "
//             << std::setw(25) << imag(vr[8*Comp::Nx+2*ix+1]) << std::endl;
//       }
//     }

    // for(int i=0; i<n; i++) std::clog << real(vr[i]) << " " << imag(vr[i]) << std::endl;
  }

  for(int i=0; i<n; i++) std::clog << i << " " << real(W[i]) << " " << imag(W[i]) << " " << abs(W[i]) << std::endl;

  /* free resources */
  free(A);
  free(h_work);

  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_W));
  CUDA_CHECK(hipFree(d_VL));
  CUDA_CHECK(hipFree(d_VR));
  CUDA_CHECK(hipFree(d_info));
  CUDA_CHECK(hipFree(d_work));

  CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
  CUSOLVER_CHECK(hipsolverDnDestroy(handle));
  CUDA_CHECK(hipStreamDestroy(stream));


  return 0; // EXIT_SUCCESS;

}

