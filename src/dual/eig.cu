#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cstdlib>
#include <cassert>
#include <algorithm>

#include <cstdint>
#include <complex>
using Idx = std::int32_t;
using Complex = std::complex<double>;

namespace CompilationConst{
  constexpr int NPARALLEL=1;

  constexpr int N_REFINE=2;
  constexpr int NS=2;
  constexpr Idx N_SITES=20*N_REFINE*N_REFINE;
  constexpr Idx N=NS*N_SITES; // matrix size of DW
}

#define IsVerbose

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hipsolver.h>
using CuC = hipDoubleComplex;
#include "gpu_header.h"

// ======================================

#include "s2n.h"
#include "rng.h"
#include "gauge.h"
#include "action.h"
#include "dirac.h"

#include "sparse_matrix.h"
#include "sparse_dirac.h"
#include "matpoly.h"
// #include "pseudofermion.h"
#include "overlap.h"

// #include "hmc.h"
// #include "dirac_s2_dual.h"
// #include "header_cusolver.hpp"


// TODO: Cusparse for SparseMatrix::act_gpu, probably defining handle in matpoly.h
// all the operation on GPU in Overlap::operator()
// gradient of Dov (Overlap class, in parallel to Dirac)
// pseudofermion
// 3d
// __m256 to vectorize with AVX2


int main(int argc, char* argv[]){
  std::cout << std::scientific << std::setprecision(15);
  std::clog << std::scientific << std::setprecision(15);


  int device;
  CUDA_CHECK(hipGetDeviceCount(&device));
  hipDeviceProp_t device_prop[device];
  hipGetDeviceProperties(&device_prop[0], 0);
  std::cout << "# dev = " << device_prop[0].name << std::endl;
  CUDA_CHECK(hipSetDevice(0));// "TITAN V"
  std::cout << "# (GPU device is set.)" << std::endl;

  // ---------------------------------------

  using Gauge=U1onS2;
  // using Force=U1onS2;
  // using Action=U1Wilson;
  // using Fermion=Dirac1fonS2;
  // using HMC=HMC<Force,Gauge,Action,Fermion>;
  using Rng=ParallelRng;

  Lattice lattice(CompilationConst::N_REFINE);
  // Dirac1fonS2 D(lattice, 0.0, 1.0);

  using WilsonDirac=Dirac1fonS2;
  // using Overlap=OverlapPseudoFermion;


  Gauge U(lattice);
  Rng rng(lattice);
  U.gaussian( rng );

  const double M5 = -2.0;
  WilsonDirac DW(lattice, M5);
  Overlap Dov(DW);
  Dov.compute(U);


  constexpr Idx N = CompilationConst::N;
  Eigen::MatrixXcd mat(N, N);
  {
    for(Idx i=0; i<N; i++){
      Eigen::VectorXcd e = Eigen::VectorXcd::Zero(N);
      e(i) = 1.0;
      std::vector<Complex> xi(e.data(), e.data()+N);
      std::vector<Complex> Dxi(N);
      Dov( Dxi, xi );
      mat.block(0,i,N,1) = Eigen::Map<Eigen::MatrixXcd>(Dxi.data(), N, 1);
    }
  }

  // =========================================
  // cusolver
  hipsolverHandle_t handle = NULL;
  hipStream_t stream = NULL;
  hipsolverDnParams_t params = NULL;

  const int n = mat.cols(); // Number of rows (or columns) of matrix A.
  const int lda = n;

  CuC *A, *W;
  A = (CuC*)malloc(n*n*CD);
  W = (CuC*)malloc(n*CD);
  for(int j=0; j<n; j++) for(int i=0; i<n; i++) A[n*j+i] = cplx(mat(i,j));
  // for(int j=0; j<n; j++) for(int i=0; i<n; i++) A[n*j+i] = reinterpret_cast<CuC*>(&mat(i,j));
  // for(int j=0; j<n; j++) for(int i=0; i<n; i++) A[n*j+i] = cplxmat(i,j));
  for(int i=0; i<n; i++) W[i] = cplx(0.);

  CuC *d_A, *d_W, *d_VL, *d_VR;
  int ldvl = n;
  int ldvr = n;
  //
  int info = 0;
  int *d_info = nullptr;
  
  size_t workspaceInBytesOnDevice = 0; /* size of workspace */
  void *d_work = nullptr;              /* device workspace */
  size_t workspaceInBytesOnHost = 0;   /* size of workspace */
  void *h_work = nullptr;              /* host workspace for */

  /* step 1: create cusolver handle, bind a stream */
  CUSOLVER_CHECK(hipsolverDnCreate(&handle));
  CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  CUSOLVER_CHECK(hipsolverSetStream(handle, stream));
  CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

  CUDA_CHECK(hipMalloc( &d_A, CD * n*n ));
  CUDA_CHECK(hipMalloc( &d_W, CD * n ));
  CUDA_CHECK(hipMalloc( &d_VL, CD * n*n ));
  CUDA_CHECK(hipMalloc( &d_VR, CD * n*n ));
  CUDA_CHECK(hipMalloc( &d_info, sizeof(int)));

  CUDA_CHECK( hipMemcpy(d_A, A, CD*n*n, H2D) );

  // step 3: query working space of syevd
  hipsolverEigMode_t jobvl = HIPSOLVER_EIG_MODE_NOVECTOR;
  hipsolverEigMode_t jobvr = HIPSOLVER_EIG_MODE_NOVECTOR;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

  CUSOLVER_CHECK( cusolverDnXgeev_bufferSize( handle,
					      params,
					      jobvl,
					      jobvr,
					      n,
					      HIP_C_64F,
					      d_A, // device
					      lda,
					      HIP_C_64F,
					      d_W, // Array holding the computed eigenvalues of A
					      HIP_C_64F,
					      d_VL,
					      ldvl,
					      HIP_C_64F,
					      d_VR,
					      ldvr,
					      HIP_C_64F,
					      &workspaceInBytesOnDevice,
					      &workspaceInBytesOnHost)
		  );

  CUDA_CHECK(hipMalloc( &d_work, workspaceInBytesOnDevice ) );
  h_work = malloc(workspaceInBytesOnHost);

  // step 4: compute spectrum
  CUSOLVER_CHECK( cusolverDnXgeev( handle,
				   params,
				   jobvl,
				   jobvr,
				   n,
				   HIP_C_64F,
				   d_A,
				   lda,
				   HIP_C_64F,
				   d_W,
				   HIP_C_64F,
				   d_VL,
				   ldvl,
				   HIP_C_64F,
				   d_VR,
				   ldvr,
				   HIP_C_64F,
				   d_work, // void *bufferOnDevice,
				   workspaceInBytesOnDevice,
				   h_work, // void *bufferOnHost,
				   workspaceInBytesOnHost,
				   d_info)
		  );

  // ---------------------------------------------

  CUDA_CHECK(hipMemcpy( W, d_W, CD*n, D2H) );
  CUDA_CHECK(hipMemcpy( &info, d_info, sizeof(int), D2H ));

  std::cout << "# info (0=success) = " << info << std::endl;
  assert( info==0 );

  // std::vector<double> res(n);
  // for(int i=0; i<n; i++) res[i] = real(W[i]);
  // std::sort(res.begin(), res.end());
  // for(int i=0; i<n; i++) std::cout << i << " "
  // 				   << res[i] << " "
  // 				   << Dov.sgn(res[i]) << std::endl;

  for(int i=0; i<n; i++) std::cout << real(W[i]) << " " << imag(W[i]) << " " << abs(W[i]) << std::endl;

  /* free resources */
  free(A);
  free(h_work);

  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_W));
  CUDA_CHECK(hipFree(d_VL));
  CUDA_CHECK(hipFree(d_VR));
  CUDA_CHECK(hipFree(d_info));
  CUDA_CHECK(hipFree(d_work));

  CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
  CUSOLVER_CHECK(hipsolverDnDestroy(handle));
  CUDA_CHECK(hipStreamDestroy(stream));


  return 0; // EXIT_SUCCESS;

  // CUDA_CHECK(hipDeviceReset());



  // // 2.4.5.7. cusolverDnXgeev()
  // hipsolverStatus_t
  //   cusolverDnXgeev_bufferSize(
  // 			       hipsolverHandle_t handle,
  // 			       hipsolverDnParams_t params,
  // 			       hipsolverEigMode_t jobvl,
  // 			       hipsolverEigMode_t jobvr,
  // 			       int64_t n,
  // 			       hipDataType dataTypeA,
  // 			       const void *A,
  // 			       int64_t lda,
  // 			       hipDataType dataTypeW,
  // 			       const void *W,
  // 			       hipDataType dataTypeVL,
  // 			       const void *VL,
  // 			       int64_t ldvl,
  // 			       hipDataType dataTypeVR,
  // 			       const void *VR,
  // 			       int64_t ldvr,
  // 			       hipDataType computeType,
  // 			       size_t *workspaceInBytesOnDevice,
  // 			       size_t *workspaceInBytesOnHost);

  // // ss. 2.5.2.5. cusolverSp<t>csreigvsi()
  // cusolverSpZcsreigvsi(hipsolverSpHandle_t handle,
  // 		       int m,
  // 		       int nnz,
  // 		       const hipsparseMatDescr_t descrA,
  // 		       const cuDoubleCuC *csrValA,
  // 		       const int *csrRowPtrA,
  // 		       const int *csrColIndA,
  // 		       cuDoubleCuC mu0,
  // 		       const cuDoubleCuC *x0,
  // 		       int maxite,
  // 		       double tol,
  // 		       cuDoubleCuC *mu,
  // 		       cuDoubleCuC *x);

  // Eigen::CuCEigenSolver<Eigen::MatrixXcd> solver( mat );
  // const Eigen::MatrixXcd evec = solver.eigenvectors();
  // Eigen::VectorXcd ev = solver.eigenvalues();
  // for(int i=0; i<evec.rows(); i++){
  //   const Eigen::VectorXcd check1 = sq * evec.col(i);
  //   const Eigen::VectorXcd check2 = eval[i] * evec.col(i);
  //   assert( (check1-check2).norm() < 1.0e-8 );

  //   const Eigen::VectorXcd MV = mat * evec.col(i);
  //   std::cout << ( MV.array() / evec.col(i).array() - 1.0).abs().maxCoeff() << std::endl;
  // }

  // auto ev = mat.eigenvalues();
  // for(int i=0; i<ev.size(); i++){
  //   std::cout << ev[i].real() << " " << ev[i].imag() << std::endl;
  // }

  // ----------------------------------

    // return 0;
}



  // for(int ix=0; ix<lattice.n_sites; ix++){
  //   for(int jj=0; jj<lattice.sites[ix].nn; jj++){
  //     const int iy = lattice.sites[ix].neighbors[jj];
  //     auto mat1 = ( D.sigma[0] - D.gamma(ix, iy) ) * D.Omega(ix, iy);
  //     auto mat2 = D.Omega(ix, iy) * ( D.sigma[0] - D.gamma(iy, ix, M_PI) );
  //     std::cout << mat1-mat2 << std::endl;
  //   }}

  // ----------------------------------
