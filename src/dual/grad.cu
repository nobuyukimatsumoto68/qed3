#include <typeinfo>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cstdlib>
#include <cassert>

#include <algorithm>


#include <cstdint>
#include <complex>
using Idx = std::int32_t;
using Complex = std::complex<double>;

namespace Comp{
  constexpr int NPARALLEL=10;
  constexpr int NSTREAMS=2;

  constexpr int N_REFINE=2;
  constexpr int NS=2;
  constexpr Idx N_SITES=20*N_REFINE*N_REFINE;
  constexpr Idx N=NS*N_SITES; // matrix size of DW

  // const double TOL=1.0e-9;
  const double TOL_INNER=1.0e-10;
  const double TOL_OUTER=1.0e-9;
}

// #define IsVerbose
// #define InfoForce
#define InfoDelta

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hipsolver.h>
using CuC = hipDoubleComplex;
#include "gpu_header.h"

// ======================================

#include "timer.h"

#include "s2n.h"
#include "rng.h"
#include "gauge.h"
#include "force.h"
#include "action.h"
#include "sparse_matrix.h"
#include "dirac.h"
#include "sparse_dirac.h"
#include "matpoly.h"
#include "overlap.h"
#include "pseudofermion.h"

# include "integrator.h"
#include "hmc.h"
// #include "dirac_s2_dual.h"
// #include "header_cusolver.hpp"


// TODO: Cusparse for SparseMatrix::act_gpu, probably defining handle in matpoly.h
// make 2 streams in V Vdag in square in Overlap
// all the operation on GPU in Overlap::operator()
// pseudofermion
// 3d
// __m256 to vectorize with AVX2


int main(int argc, char* argv[]){
  std::cout << std::scientific << std::setprecision(15);
  std::clog << std::scientific << std::setprecision(15);

  int device;
  CUDA_CHECK(hipGetDeviceCount(&device));
  hipDeviceProp_t device_prop[device];
  hipGetDeviceProperties(&device_prop[0], 0);
  std::cout << "# dev = " << device_prop[0].name << std::endl;
  CUDA_CHECK(hipSetDevice(0));// "TITAN V"
  std::cout << "# (GPU device is set.)" << std::endl;

  // ---------------------------------------

  using Lattice=S2Trivalent;
  using Gauge=U1onS2<false>;
  using Force=U1onS2<false>;
  using Action=U1Wilson;
  using Fermion=Overlap;
  using Rng=ParallelRng<Lattice>;
  using WilsonDirac=Dirac1fonS2;

  using Link = std::array<Idx,2>; // <int,int>;
  constexpr Idx N = Comp::N;

  // ----------------------

  Lattice lattice(Comp::N_REFINE);
  Gauge U(lattice);
  Rng rng(lattice);
  U.gaussian( rng, 0.2 );

  // ------------------

  const double gR = 0.4;
  const double beta = 1.0/(gR*gR);
  Action SW(beta);

  // -----------------

  const double M5 = -1.8;
  WilsonDirac DW(lattice, M5);

  Fermion Dov(DW, 11);
  // Fermion Dov(DW, 5);

  const auto f_DHDov = std::bind(&Overlap::sq_deviceAsyncLaunch, &Dov,
                                 std::placeholders::_1, std::placeholders::_2);
  LinOpWrapper M_DHDov( f_DHDov );
  // const auto f_DHDov = std::bind(&Overlap::sq_device, &Dov,
  //                                std::placeholders::_1, std::placeholders::_2);
  // LinOpWrapper M_DHDov( f_DHDov );
  MatPoly Op_DHDov; Op_DHDov.push_back ( cplx(1.0), {&M_DHDov} );
  // auto f_DHov = std::bind(&Overlap::adj_device, &Dov,
  //                         std::placeholders::_1, std::placeholders::_2);
  auto f_DHov = std::bind(&Overlap::adj_deviceAsyncLaunch, &Dov,
                          std::placeholders::_1, std::placeholders::_2);

  // auto f_mgrad_DHDov = std::bind(&Overlap::grad_device, &Dov,
  //                                std::placeholders::_1, std::placeholders::_2, std::placeholders::_3);
  auto f_mgrad_DHDov = std::bind(&Overlap::grad_deviceAsyncLaunch, &Dov,
                                 std::placeholders::_1, std::placeholders::_2, std::placeholders::_3);

  PseudoFermion pf( Op_DHDov, f_DHov, f_mgrad_DHDov );


  // ------------------

  // Idx il=2;
  // Link ell = lattice.links[il];

  // const double eps = 1.0e-5;
  // Gauge UP(U);
  // UP[il] += eps;
  // Gauge UM(U);
  // UM[il] -= eps;

  // std::cout << " --- Dov.update : " << timer.currentSeconds() << std::endl;
  // Dov.update(U);
  // std::cout << " --- pf.gen : " << timer.currentSeconds() << std::endl;
  // pf.gen( rng );

  // std::cout << " --- grad constructor : " << timer.currentSeconds() << std::endl;
  // Force grad(lattice);

  // std::cout << " --- get force : " << timer.currentSeconds() << std::endl;
  // Dov.precalc_grad_deviceAsyncLaunch( U, pf.d_eta );
  // pf.get_force( grad, U );

  // std::cout << " --- fin : " << timer.currentSeconds() << std::endl;

  // std::cout << "grad = " << grad[il] << std::endl;
  // Dov.update(UP);
  // pf.update_eta();
  // double sfp = pf.S();

  // Dov.update(UM);
  // pf.update_eta();
  // double sfm = pf.S();

  // double chck = (sfp-sfm)/(2.0*eps);
  // std::cout << "check = " << chck << std::endl;

  // -----------------

  Force pi( lattice );
  pi.gaussian( rng );
  Force pi0=pi;

  Gauge U0=U;
  Dov.update(U);
  pf.gen( rng );
  Dov.precalc_grad_deviceAsyncLaunch( U, pf.d_eta );

  double tmax = 1.0; // 0.1
  // for(int nsteps=1; nsteps<=1; nsteps+=1){
  const int nsteps=5;
  // ExplicitLeapfrog integrator( tmax, nsteps );
  ExplicitLeapfrogML integrator( tmax, nsteps, 100 );
  HMC hmc(rng, &SW, &Dov, U, pi, &pf, &integrator);
  // pi = pi0;
  // U = U0;
  // Dov.update( U ); pf.update_eta();
  //     const double h0 = hmc.H();
  //   hmc.integrate();
  //   const double h1 = hmc.H();
  //   double dH = h1-h0;
  //   std::cout << tmax/nsteps << " " << dH << std::endl;
  // }

  double r, dH;
  bool is_accept;
  for(int k=0; k<20; k++){
    Timer timer;
    hmc.run( r, dH, is_accept, true);
    std::cout << "# dH : " << dH
              << " is_accept : " << is_accept << std::endl;
    std::cout << "# HMC : " << timer.currentSeconds() << " sec" << std::endl;
  }

  double r_mean;
  const int kmax=50;
  for(int k=0; k<kmax; k++){
    Timer timer;
    hmc.run( r, dH, is_accept);
    std::cout << "# dH : " << dH
              << " is_accept : " << is_accept << std::endl;
    r_mean += r;
    std::cout << "# HMC : " << timer.currentSeconds() << " sec" << std::endl;
  }
  r_mean /= kmax;
  std::cout << "# r_mean = " << r_mean << std::endl;



  // CUDA_CHECK(hipDeviceReset());
  return 0;

}

