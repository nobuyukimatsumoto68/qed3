#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cstdlib>
#include <cassert>
#include <algorithm>

#include <cstdint>
#include <complex>
using Idx = std::int32_t;
using Complex = std::complex<double>;

namespace CompilationConst{
  constexpr int NPARALLEL=10;

  constexpr int N_REFINE=2;
  constexpr int NS=2;
  constexpr Idx N_SITES=20*N_REFINE*N_REFINE;
  constexpr Idx N=NS*N_SITES; // matrix size of DW
}

#define IsVerbose

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hipsolver.h>
using CuC = hipDoubleComplex;
#include "gpu_header.h"

// ======================================

#include "s2n.h"
#include "rng.h"
#include "gauge.h"
#include "action.h"

#include "sparse_matrix.h"
// #include "pseudofermion.h"

#include "dirac.h"

#include "sparse_dirac.h"
#include "matpoly.h"

#include "overlap.h"

// #include "hmc.h"
// #include "dirac_s2_dual.h"
// #include "header_cusolver.hpp"


// TODO: Cusparse for SparseMatrix::act_gpu, probably defining handle in matpoly.h
// all the operation on GPU in Overlap::operator()
// gradient of Dov (Overlap class, in parallel to Dirac)
// pseudofermion
// 3d
// __m256 to vectorize with AVX2


int main(int argc, char* argv[]){
  std::cout << std::scientific << std::setprecision(15);
  std::clog << std::scientific << std::setprecision(15);


  int device;
  CUDA_CHECK(hipGetDeviceCount(&device));
  hipDeviceProp_t device_prop[device];
  hipGetDeviceProperties(&device_prop[0], 0);
  std::cout << "# dev = " << device_prop[0].name << std::endl;
  CUDA_CHECK(hipSetDevice(0));// "TITAN V"
  std::cout << "# (GPU device is set.)" << std::endl;

  // ---------------------------------------

  using Gauge=U1onS2;
  // using Force=U1onS2;
  // using Action=U1Wilson;
  // using Fermion=Dirac1fonS2;
  // using HMC=HMC<Force,Gauge,Action,Fermion>;
  using Rng=ParallelRng;

  Lattice lattice(CompilationConst::N_REFINE);
  // Dirac1fonS2 D(lattice, 0.0, 1.0);

  using WilsonDirac=Dirac1fonS2;
  // using Overlap=OverlapPseudoFermion;

  Gauge U(lattice);
  Rng rng(lattice);
  // U.gaussian( rng );

  const double M5 = -2.0;
  WilsonDirac DW(lattice, M5);

  constexpr Idx N = CompilationConst::N;

  using Link = std::array<Idx,2>; // <int,int>;

  // const Idx ix=2;
  // const Idx iy=lattice.nns[ix][0];
  // const Link ell{ix,iy};

  // COO coo;
  // DW.d_coo_format(coo.en, U, ell);
  // coo.do_it();








  // {
  //   // PseudoFermion phi( D, U, rng );
  //   // Force exact = phi.dS( U );

  //   Overlap Dov(DW);
  //   Dov.compute(U);

  //   std::vector<Complex> xi(N);
  //   for(int i=0; i<N; i++) xi[i] = rng.gaussian();

  //   // -------------

  //   std::vector<Complex> eta(N);
  //   {
  //     MatPoly Op;
  //     Op.push_back ( cplx(1.0), {&(Dov.M_DW), &(Dov.M_DWH)} );
  //     Op.solve<N>( eta, xi );
  //   }

  //   double Sf = 0.0;
  //   for(Idx i=0; i<N; i++) Sf += std::real( std::conj(xi[i]) * eta[i] );

  //   // -------------

  //   Idx il=0;

  //   {
  //     double dSf = Dov.grad( lattice.links[il], U, eta );
  //     std::cout << "grad = " << dSf << std::endl;
  //   }

  //   // -------------

  //   {
  //     double Sfp = 0.0, Sfm = 0.0;
  //     const double eps = 1.0e-5;

  //     {
  //       Gauge UP(U);
  //       UP[il] += eps;
  //       Dov.compute(UP);

  //       MatPoly Op;
  //       Op.push_back ( cplx(1.0), {&(Dov.M_DW), &(Dov.M_DWH)} );
  //       Op.solve<N>( eta, xi );
  //       for(Idx i=0; i<N; i++) Sfp += std::real( std::conj(xi[i]) * eta[i] );
  //     }

  //     {
  //       Gauge UM(U);
  //       UM[il] -= eps;
  //       Dov.compute(UM);

  //       MatPoly Op;
  //       Op.push_back ( cplx(1.0), {&(Dov.M_DW), &(Dov.M_DWH)} );
  //       Op.solve<N>( eta, xi );
  //       for(Idx i=0; i<N; i++) Sfm += std::real( std::conj(xi[i]) * eta[i] );
  //     }

  //     std::cout << (Sfp-Sfm)/(2.0*eps) << std::endl;
  //   }

  // }



















    // -------------

    // std::vector<Complex> deriv(N);
    // // for(int il=0; il<U.lattice.n_links; il++){
    // {
    //   Idx il=0;

    //   const double eps = 1.0e-5;
    //   Gauge UP(U);
    //   Gauge UM(U);

    //   UP[il] += eps;
    //   UM[il] -= eps;

    //   std::vector<Complex> DxiP(N), DxiM(N);
    //   Dov.compute(UP);
    //   Dov( DxiP, xi );
    //   Dov.compute(UM);
    //   Dov( DxiM, xi );

    //   for(int i=0; i<N; i++) deriv[i] = (DxiP[i]-DxiM[i])/(2.0*eps);

    //   // double numeric = ( phi.S(UP) - phi.S(UM) ) / (2.0*eps);
    //   for(int i=0; i<N; i++) {
    //     std::cout << deriv[i] << std::endl;
    //   }
    // }






  // Eigen::MatrixXcd mat(N, N);
  // {
  //   for(Idx i=0; i<N; i++){
  //     Eigen::VectorXcd e = Eigen::VectorXcd::Zero(N);
  //     e(i) = 1.0;
  //     std::vector<Complex> xi(e.data(), e.data()+N);
  //     std::vector<Complex> Dxi(N);
  //     Dov( Dxi, xi );
  //     mat.block(0,i,N,1) = Eigen::Map<Eigen::MatrixXcd>(Dxi.data(), N, 1);
  //   }
  // }
  // std::cout << Dov.lambda_max << std::endl;


  // return 0; // EXIT_SUCCESS;

  // CUDA_CHECK(hipDeviceReset());



  // // 2.4.5.7. cusolverDnXgeev()
  // hipsolverStatus_t
  //   cusolverDnXgeev_bufferSize(
  // 			       hipsolverHandle_t handle,
  // 			       hipsolverDnParams_t params,
  // 			       hipsolverEigMode_t jobvl,
  // 			       hipsolverEigMode_t jobvr,
  // 			       int64_t n,
  // 			       hipDataType dataTypeA,
  // 			       const void *A,
  // 			       int64_t lda,
  // 			       hipDataType dataTypeW,
  // 			       const void *W,
  // 			       hipDataType dataTypeVL,
  // 			       const void *VL,
  // 			       int64_t ldvl,
  // 			       hipDataType dataTypeVR,
  // 			       const void *VR,
  // 			       int64_t ldvr,
  // 			       hipDataType computeType,
  // 			       size_t *workspaceInBytesOnDevice,
  // 			       size_t *workspaceInBytesOnHost);

  // // ss. 2.5.2.5. cusolverSp<t>csreigvsi()
  // cusolverSpZcsreigvsi(hipsolverSpHandle_t handle,
  // 		       int m,
  // 		       int nnz,
  // 		       const hipsparseMatDescr_t descrA,
  // 		       const cuDoubleCuC *csrValA,
  // 		       const int *csrRowPtrA,
  // 		       const int *csrColIndA,
  // 		       cuDoubleCuC mu0,
  // 		       const cuDoubleCuC *x0,
  // 		       int maxite,
  // 		       double tol,
  // 		       cuDoubleCuC *mu,
  // 		       cuDoubleCuC *x);

  // Eigen::CuCEigenSolver<Eigen::MatrixXcd> solver( mat );
  // const Eigen::MatrixXcd evec = solver.eigenvectors();
  // Eigen::VectorXcd ev = solver.eigenvalues();
  // for(int i=0; i<evec.rows(); i++){
  //   const Eigen::VectorXcd check1 = sq * evec.col(i);
  //   const Eigen::VectorXcd check2 = eval[i] * evec.col(i);
  //   assert( (check1-check2).norm() < 1.0e-8 );

  //   const Eigen::VectorXcd MV = mat * evec.col(i);
  //   std::cout << ( MV.array() / evec.col(i).array() - 1.0).abs().maxCoeff() << std::endl;
  // }

  // auto ev = mat.eigenvalues();
  // for(int i=0; i<ev.size(); i++){
  //   std::cout << ev[i].real() << " " << ev[i].imag() << std::endl;
  // }

  // ----------------------------------

    // return 0;
}



  // for(int ix=0; ix<lattice.n_sites; ix++){
  //   for(int jj=0; jj<lattice.sites[ix].nn; jj++){
  //     const int iy = lattice.sites[ix].neighbors[jj];
  //     auto mat1 = ( D.sigma[0] - D.gamma(ix, iy) ) * D.Omega(ix, iy);
  //     auto mat2 = D.Omega(ix, iy) * ( D.sigma[0] - D.gamma(iy, ix, M_PI) );
  //     std::cout << mat1-mat2 << std::endl;
  //   }}

  // ----------------------------------
